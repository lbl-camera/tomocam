#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cusp/complex.h>
#include <cusp/blas.h>
#include<cusp/csr_matrix.h>
#include<cusp/multiply.h>
#include <cusp/array1d.h>
#include <cusp/copy.h>
#include <thrust/device_ptr.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"

/* Input Arguments */
#define	VAL	prhs[0]
#define	COL	prhs[1]
#define	ROW	prhs[2]
// #define	NCOL    prhs[3]
 #define	NROW    prhs[3]
// #define	NNZ    prhs[5]
#define	XV    prhs[4]


/* Output Arguments */
#define	Y	plhs[0]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs,const mxArray * prhs[]){
    
    mxGPUArray const *Aval;
    mxGPUArray const *Acol;
    mxGPUArray const *Arow;
    mxGPUArray const *x;
    mxGPUArray  *y;
    
//     int nnzs = lrint(mxGetScalar(NCOL));
     int nrow = lrint(mxGetScalar(NROW));
//       mexPrintf("nrows=%d", nrow);
//     int nptr=nrows+1;
//     int nnz  = lrint(mxGetScalar(NNZ));
//     
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    
    /*get matlab variables*/
    Aval = mxGPUCreateFromMxArray(VAL);
    Acol = mxGPUCreateFromMxArray(COL);
    Arow = mxGPUCreateFromMxArray(ROW);
    x    = mxGPUCreateFromMxArray(XV);
    
    int nnz=mxGPUGetNumberOfElements(Acol);
//    int nrow=mxGPUGetNumberOfElements(Arow);
    int ncol =mxGPUGetNumberOfElements(x);

    
    mxComplexity isXVreal = mxGPUGetComplexity(x);
    mxComplexity isAreal = mxGPUGetComplexity(Aval);
    const mwSize ndim= 1;
    const mwSize dims[]={(mwSize) (nrow)};

    if (isAreal!=isXVreal)
    {
        mexErrMsgTxt("Aval and X must have the same complexity");
        return;
    }

//    if(mxGPUGetClassID(Aval) != mxSINGLE_CLASS||
 //  mxGPUGetClassID(x)!= mxSINGLE_CLASS||
 //  mxGPUGetClassID(Arow)!= mxINT32_CLASS||
 //  mxGPUGetClassID(Acol)!= mxINT32_CLASS){
  //   mexErrMsgTxt("usage: gspmv(single, int32, int32, single )");
  //   return;
  //  }
    
    
    // single or double
    if(((mxGPUGetClassID(Aval) != mxSINGLE_CLASS || mxGPUGetClassID(x)!= mxSINGLE_CLASS) &&
            ((mxGPUGetClassID(Aval) != mxDOUBLE_CLASS) || mxGPUGetClassID(x)!= mxDOUBLE_CLASS))||
            mxGPUGetClassID(Arow)!= mxINT32_CLASS||    mxGPUGetClassID(Acol)!= mxINT32_CLASS){
        mexErrMsgTxt("usage: gspmv(single/double, int32, int32, single/double )");
        return;
    };
    
    
    //create output vector
//     mexPrintf("\ncreating nrows=%d", dims[0]);
    y = mxGPUCreateGPUArray(ndim,dims,mxGPUGetClassID(x),isAreal, MX_GPU_DO_NOT_INITIALIZE);
     
    
    /* wrap indices from matlab */
    typedef const int  TI;  /* the type for index */
    TI *d_col =(TI  *)(mxGPUGetDataReadOnly(Acol));
    TI *d_row =(TI  *)(mxGPUGetDataReadOnly(Arow));
    // wrap with thrust::device_ptr
    thrust::device_ptr<TI>    wrap_d_col  (d_col);
    thrust::device_ptr<TI>    wrap_d_row  (d_row);
    // wrap with array1d_view 
    typedef typename cusp::array1d_view< thrust::device_ptr<TI> >   idx2Av;
    // wrap index arrays
    idx2Av colIndex (wrap_d_col , wrap_d_col + nnz);
    idx2Av rowIndex (wrap_d_row , wrap_d_row + nnz);
           
    if (isAreal!=mxREAL){
                 if (mxGPUGetClassID(Aval) != mxSINGLE_CLASS)
                 {
                         typedef const cusp::complex<double> TA;  /* the type for A */
        typedef const cusp::complex<double> TXV; /* the type for X */
        typedef cusp::complex<double> TYV; /* the type for Y */

        // wrap with array1d_view 
        typedef typename cusp::array1d_view< thrust::device_ptr<TA > >   val2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TXV > >   x2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TYV > >   y2Av;
        
        /* pointers from matlab */
        TA *d_val =(TA  *)(mxGPUGetDataReadOnly(Aval));
        TXV *d_x   =(TXV  *)(mxGPUGetDataReadOnly(x));
        TYV *d_y   =(TYV  *)(mxGPUGetData(y));
        
        // wrap with thrust::device_ptr
        thrust::device_ptr<TA >    wrap_d_val  (d_val);
        thrust::device_ptr<TXV >    wrap_d_x    (d_x);
        thrust::device_ptr<TYV >    wrap_d_y  (d_y);
        
        // wrap  arrays
        val2Av valIndex (wrap_d_val , wrap_d_val + nnz);
        x2Av xIndex   (wrap_d_x   , wrap_d_x   + ncol);
        y2Av yIndex(wrap_d_y, wrap_d_y+ nrow);
//        y2Av yIndex(wrap_d_y, wrap_d_y+ ncol);
        
        // combine info in CSR matrix
        typedef  cusp::coo_matrix_view<idx2Av,idx2Av,val2Av> DeviceView;
        
        DeviceView As(nrow, ncol, nnz, rowIndex, colIndex, valIndex);
                
        // multiply matrix
        cusp::multiply(As, xIndex, yIndex);
                 
                 }else{
        typedef const cusp::complex<float> TA;  /* the type for A */
        typedef const cusp::complex<float> TXV; /* the type for X */
        typedef cusp::complex<float> TYV; /* the type for Y */

        // wrap with array1d_view 
        typedef typename cusp::array1d_view< thrust::device_ptr<TA > >   val2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TXV > >   x2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TYV > >   y2Av;
        
        /* pointers from matlab */
        TA *d_val =(TA  *)(mxGPUGetDataReadOnly(Aval));
        TXV *d_x   =(TXV  *)(mxGPUGetDataReadOnly(x));
        TYV *d_y   =(TYV  *)(mxGPUGetData(y));
        
        // wrap with thrust::device_ptr
        thrust::device_ptr<TA >    wrap_d_val  (d_val);
        thrust::device_ptr<TXV >    wrap_d_x    (d_x);
        thrust::device_ptr<TYV >    wrap_d_y  (d_y);
        
        // wrap  arrays
        val2Av valIndex (wrap_d_val , wrap_d_val + nnz);
        x2Av xIndex   (wrap_d_x   , wrap_d_x   + ncol);
        y2Av yIndex(wrap_d_y, wrap_d_y+ nrow);
//        y2Av yIndex(wrap_d_y, wrap_d_y+ ncol);
        
        // combine info in CSR matrix
        typedef  cusp::coo_matrix_view<idx2Av,idx2Av,val2Av> DeviceView;
        
        DeviceView As(nrow, ncol, nnz, rowIndex, colIndex, valIndex);
                
        // multiply matrix
        cusp::multiply(As, xIndex, yIndex);
                 }
    }
     else{
                 if (mxGPUGetClassID(Aval) != mxSINGLE_CLASS)
                 {
                             typedef const double TA;  /* the type for A */
        typedef const double TXV; /* the type for X */
        typedef double TYV; /* the type for Y */
   
        /* pointers from matlab */
        TA *d_val =(TA  *)(mxGPUGetDataReadOnly(Aval));
        TXV *d_x   =(TXV  *)(mxGPUGetDataReadOnly(x));
        TYV *d_y   =(TYV  *)(mxGPUGetData(y));
        
        // wrap with thrust::device_ptr!
        thrust::device_ptr<TA >    wrap_d_val  (d_val);
        thrust::device_ptr<TXV >    wrap_d_x    (d_x);
        thrust::device_ptr<TYV >    wrap_d_y  (d_y);
        // wrap with array1d_view 
        typedef typename cusp::array1d_view< thrust::device_ptr<TA > >   val2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TXV > >   x2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TYV > >   y2Av;
        
        // wrap  arrays
        val2Av valIndex (wrap_d_val , wrap_d_val + nnz);
        x2Av xIndex   (wrap_d_x   , wrap_d_x   + ncol);
        //y2Av yIndex(wrap_d_y, wrap_d_y+ ncol);        
        y2Av yIndex(wrap_d_y, wrap_d_y+ nrow);
        
        // combine info in CSR matrix
        typedef  cusp::csr_matrix_view<idx2Av,idx2Av,val2Av> DeviceView;
        
        DeviceView As(nrow, ncol, nnz, rowIndex, colIndex, valIndex);
                
        // multiply matrix
        cusp::multiply(As, xIndex, yIndex);

        }else{ //real single precision
         
        typedef const float TA;  /* the type for A */
        typedef const float TXV; /* the type for X */
        typedef float TYV; /* the type for Y */
   
        /* pointers from matlab */
        TA *d_val =(TA  *)(mxGPUGetDataReadOnly(Aval));
        TXV *d_x   =(TXV  *)(mxGPUGetDataReadOnly(x));
        TYV *d_y   =(TYV  *)(mxGPUGetData(y));
        
        // wrap with thrust::device_ptr!
        thrust::device_ptr<TA >    wrap_d_val  (d_val);
        thrust::device_ptr<TXV >    wrap_d_x    (d_x);
        thrust::device_ptr<TYV >    wrap_d_y  (d_y);
        // wrap with array1d_view 
        typedef typename cusp::array1d_view< thrust::device_ptr<TA > >   val2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TXV > >   x2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TYV > >   y2Av;
        
        // wrap  arrays
        val2Av valIndex (wrap_d_val , wrap_d_val + nnz);
        x2Av xIndex   (wrap_d_x   , wrap_d_x   + ncol);
        //y2Av yIndex(wrap_d_y, wrap_d_y+ ncol);        
        y2Av yIndex(wrap_d_y, wrap_d_y+ nrow);
        
        // combine info in CSR matrix
        typedef  cusp::csr_matrix_view<idx2Av,idx2Av,val2Av> DeviceView;
        
        DeviceView As(nrow, ncol, nnz, rowIndex, colIndex, valIndex);
                
        // multiply matrix
        cusp::multiply(As, xIndex, yIndex);
        
    }
     }
    Y = mxGPUCreateMxArrayOnGPU(y);
    
    mxGPUDestroyGPUArray(Aval);
    mxGPUDestroyGPUArray(Arow);
    mxGPUDestroyGPUArray(Acol);
    mxGPUDestroyGPUArray(x);
    mxGPUDestroyGPUArray(y);

    return;
}

