#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <cusp/blas.h>
#include<cusp/csr_matrix.h>
#include<cusp/multiply.h>
#include <cusp/array1d.h>
#include <cusp/copy.h>
#include <thrust/device_ptr.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"


/* Input Arguments */
#define	ROW	prhs[0]
#define	NPTR    prhs[1]
#define	NNZ    prhs[2]

/* Output Arguments */
#define	ROW_OUT	plhs[0]




void mexFunction(int nlhs, mxArray * plhs[], int nrhs,const mxArray * prhs[]){
    mxGPUArray const *Arow;
    mxGPUArray  *rowptr;
    mxInitGPU();     /* Initialize the MathWorks GPU API. */
   int nptr = lrint(mxGetScalar(NPTR));
    int nnz  = lrint(mxGetScalar(NNZ));
    const mwSize ndim= 1;    
//    const mwSize dimcol[]={mwSize(nnz)};    
     const mwSize dimptr[]={mwSize(nptr)};    
//      mexPrintf("nrows=%d,nnz=%d\n", dimptr[0],dimcol[0]);

    // input output array 
    Arow = mxGPUCreateFromMxArray(ROW);
      rowptr = mxGPUCreateGPUArray(ndim,dimptr,mxINT32_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
//          mexPrintf("created matrix, nrows=%d,nnz=%d\n", nptr,nnz);
    // pointer from matlab
    int *d_Arow =(int  *)(mxGPUGetDataReadOnly(Arow));    
    int *d_rowptr =(int  *)(mxGPUGetData(rowptr));

    // wrap with thrust::device_ptr
    thrust::device_ptr<int>    wrap_d_Arow  (d_Arow);
    thrust::device_ptr<int>    wrap_d_rowptr  (d_rowptr);
    // convert to ptr 
      thrust::lower_bound(wrap_d_Arow,
                        wrap_d_Arow+nnz,
                        thrust::counting_iterator<int>(0),
                        thrust::counting_iterator<int>(nptr),
                        wrap_d_rowptr);
    //bring back to matlab
    ROW_OUT = mxGPUCreateMxArrayOnGPU(rowptr);
    //clean up
    mxGPUDestroyGPUArray(Arow);
    mxGPUDestroyGPUArray(rowptr);

    return;
}

