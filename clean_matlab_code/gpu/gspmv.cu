#include <hip/hip_runtime.h>
#include <cusp/complex.h>
#include <cusp/blas.h>
#include<cusp/csr_matrix.h>
#include<cusp/multiply.h>
#include <cusp/array1d.h>
#include <cusp/copy.h>
#include <thrust/device_ptr.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"


/* Input Arguments */
#define	VAL	prhs[0]
#define	COL	prhs[1]
#define	PTR	prhs[2]
#define	XV    prhs[3]

/* Output Arguments */
#define	Y	plhs[0]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs,const mxArray * prhs[]){
    
    mxGPUArray const *Aval;
    mxGPUArray const *Acol;
    mxGPUArray const *Aptr;
    mxGPUArray const *x;
    mxGPUArray  *y;
    
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    
    /*get matlab variables*/
    Aval = mxGPUCreateFromMxArray(VAL);
    Acol = mxGPUCreateFromMxArray(COL);
    Aptr = mxGPUCreateFromMxArray(PTR);
    x    = mxGPUCreateFromMxArray(XV);
    
    int ncol=mxGPUGetNumberOfElements(Acol);
    int nrowp1=mxGPUGetNumberOfElements(Aptr);
    int nin =mxGPUGetNumberOfElements(x);

//    int nout=nrowp1-1;
    
    mxComplexity isXVreal = mxGPUGetComplexity(x);
    mxComplexity isAreal = mxGPUGetComplexity(Aval);
    const mwSize ndim= 1;
    const mwSize dims[]={(mwSize) (nrowp1-1)};
//    mxComplexity isYVreal=mxCOMPLEX;
//     if (isAreal==mxREAL && isXVreal==mxREAL) 
//         isYVreal=mxCOMPLEX;

            
    if (isAreal!=isXVreal)
    {
     mexErrMsgTxt("Aval and X must have the same complexity");
    return;
    }
    
    if((mxGPUGetClassID(Aval) != mxSINGLE_CLASS && mxGPUGetClassID(Aval) != mxDOUBLE_CLASS)||
   (mxGPUGetClassID(x)!= mxSINGLE_CLASS && mxGPUGetClassID(x)!= mxDOUBLE_CLASS )||
   mxGPUGetClassID(Aptr)!= mxINT32_CLASS||    mxGPUGetClassID(Acol)!= mxINT32_CLASS){
     mexErrMsgTxt("usage: gspmv(single/double, int32, int32, single/double )");
     return;
    }
//    if (mxGPUGetClassID(Aval) != mxSINGLE_CLASS || mxGPUGetClassID(x)!= mxSINGLE_CLASS) && (mxGPUGetClassID(Aval) != mxDOUBLE_CLASS || mxGPUGetClassID(x)!= mxDOUBLE_CLASS)
    
    //create output vector
    y = mxGPUCreateGPUArray(ndim,dims,mxGPUGetClassID(x),isAreal, MX_GPU_DO_NOT_INITIALIZE);
     
    
    /* wrap indices from matlab */
    typedef const int  TI;  /* the type for index */
    TI *d_col =(TI  *)(mxGPUGetDataReadOnly(Acol));
    TI *d_ptr =(TI  *)(mxGPUGetDataReadOnly(Aptr));
    // wrap with thrust::device_ptr
    thrust::device_ptr<TI>    wrap_d_col  (d_col);
    thrust::device_ptr<TI>    wrap_d_ptr  (d_ptr);
    // wrap with array1d_view 
    typedef typename cusp::array1d_view< thrust::device_ptr<TI> >   idx2Av;
    // wrap index arrays
    idx2Av colIndex (wrap_d_col , wrap_d_col + ncol);
    idx2Av ptrIndex (wrap_d_ptr , wrap_d_ptr + nrowp1);
           
    if (isAreal!=mxREAL){

        if (mxGPUGetClassID(Aval) != mxSINGLE_CLASS)
        {                    
        typedef const cusp::complex<float> TA;  /* the type for A */
        typedef const cusp::complex<float> TXV; /* the type for X */
        typedef cusp::complex<float> TYV; /* the type for Y */
        
        
        // wrap with array1d_view 
        typedef typename cusp::array1d_view< thrust::device_ptr<TA > >   val2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TXV > >   x2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TYV > >   y2Av;
        
        /* pointers from matlab */
        TA *d_val =(TA  *)(mxGPUGetDataReadOnly(Aval));
        TXV *d_x   =(TXV  *)(mxGPUGetDataReadOnly(x));
        TYV *d_y   =(TYV  *)(mxGPUGetData(y));
        
        // wrap with thrust::device_ptr
        thrust::device_ptr<TA >    wrap_d_val  (d_val);
        thrust::device_ptr<TXV >    wrap_d_x    (d_x);
        thrust::device_ptr<TYV >    wrap_d_y  (d_y);
        
        // wrap  arrays
        val2Av valIndex (wrap_d_val , wrap_d_val + ncol);
        x2Av xIndex   (wrap_d_x   , wrap_d_x   + nin);
        y2Av yIndex(wrap_d_y, wrap_d_y+ nrowp1-1);
//        y2Av yIndex(wrap_d_y, wrap_d_y+ nin);
        
        // combine info in CSR matrix
        typedef  cusp::csr_matrix_view<idx2Av,idx2Av,val2Av> DeviceView;
        
        DeviceView As(nrowp1-1, nin, ncol, ptrIndex, colIndex, valIndex);
                
        // multiply matrix
        cusp::multiply(As, xIndex, yIndex);
        
        }else{
        
        typedef const cusp::complex<float> TA;  /* the type for A */
        typedef const cusp::complex<float> TXV; /* the type for X */
        typedef cusp::complex<float> TYV; /* the type for Y */

        // wrap with array1d_view 
        typedef typename cusp::array1d_view< thrust::device_ptr<TA > >   val2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TXV > >   x2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TYV > >   y2Av;
        
        /* pointers from matlab */
        TA *d_val =(TA  *)(mxGPUGetDataReadOnly(Aval));
        TXV *d_x   =(TXV  *)(mxGPUGetDataReadOnly(x));
        TYV *d_y   =(TYV  *)(mxGPUGetData(y));
        
        // wrap with thrust::device_ptr
        thrust::device_ptr<TA >    wrap_d_val  (d_val);
        thrust::device_ptr<TXV >    wrap_d_x    (d_x);
        thrust::device_ptr<TYV >    wrap_d_y  (d_y);
        
        // wrap  arrays
        val2Av valIndex (wrap_d_val , wrap_d_val + ncol);
        x2Av xIndex   (wrap_d_x   , wrap_d_x   + nin);
        y2Av yIndex(wrap_d_y, wrap_d_y+ nrowp1-1);
//        y2Av yIndex(wrap_d_y, wrap_d_y+ nin);
        
        // combine info in CSR matrix
        typedef  cusp::csr_matrix_view<idx2Av,idx2Av,val2Av> DeviceView;
        
        DeviceView As(nrowp1-1, nin, ncol, ptrIndex, colIndex, valIndex);
                
        // multiply matrix
        cusp::multiply(As, xIndex, yIndex);
        
    }} else{

                 if (mxGPUGetClassID(Aval) != mxSINGLE_CLASS)
                 {
                             typedef const double TA;  /* the type for A */
        typedef const double TXV; /* the type for X */
        typedef double TYV; /* the type for Y */
   
        /* pointers from matlab */
        TA *d_val =(TA  *)(mxGPUGetDataReadOnly(Aval));
        TXV *d_x   =(TXV  *)(mxGPUGetDataReadOnly(x));
        TYV *d_y   =(TYV  *)(mxGPUGetData(y));
        
        // wrap with thrust::device_ptr!
        thrust::device_ptr<TA >    wrap_d_val  (d_val);
        thrust::device_ptr<TXV >    wrap_d_x    (d_x);
        thrust::device_ptr<TYV >    wrap_d_y  (d_y);
        // wrap with array1d_view 
        typedef typename cusp::array1d_view< thrust::device_ptr<TA > >   val2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TXV > >   x2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TYV > >   y2Av;
        
        // wrap  arrays
        val2Av valIndex (wrap_d_val , wrap_d_val + ncol);
        x2Av xIndex   (wrap_d_x   , wrap_d_x   + nin);
        //y2Av yIndex(wrap_d_y, wrap_d_y+ nin);        
        y2Av yIndex(wrap_d_y, wrap_d_y+ nrowp1-1);
        
        // combine info in CSR matrix
        typedef  cusp::csr_matrix_view<idx2Av,idx2Av,val2Av> DeviceView;
        
        DeviceView As(nrowp1-1, nin, ncol, ptrIndex, colIndex, valIndex);
                
        // multiply matrix
        cusp::multiply(As, xIndex, yIndex);

                 }else{
        typedef const float TA;  /* the type for A */
        typedef const float TXV; /* the type for X */
        typedef float TYV; /* the type for Y */
   
        /* pointers from matlab */
        TA *d_val =(TA  *)(mxGPUGetDataReadOnly(Aval));
        TXV *d_x   =(TXV  *)(mxGPUGetDataReadOnly(x));
        TYV *d_y   =(TYV  *)(mxGPUGetData(y));
        
        // wrap with thrust::device_ptr!
        thrust::device_ptr<TA >    wrap_d_val  (d_val);
        thrust::device_ptr<TXV >    wrap_d_x    (d_x);
        thrust::device_ptr<TYV >    wrap_d_y  (d_y);
        // wrap with array1d_view 
        typedef typename cusp::array1d_view< thrust::device_ptr<TA > >   val2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TXV > >   x2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TYV > >   y2Av;
        
        // wrap  arrays
        val2Av valIndex (wrap_d_val , wrap_d_val + ncol);
        x2Av xIndex   (wrap_d_x   , wrap_d_x   + nin);
        //y2Av yIndex(wrap_d_y, wrap_d_y+ nin);        
        y2Av yIndex(wrap_d_y, wrap_d_y+ nrowp1-1);
        
        // combine info in CSR matrix
        typedef  cusp::csr_matrix_view<idx2Av,idx2Av,val2Av> DeviceView;
        
        DeviceView As(nrowp1-1, nin, ncol, ptrIndex, colIndex, valIndex);
                
        // multiply matrix
        cusp::multiply(As, xIndex, yIndex);
                 }
    }

    Y = mxGPUCreateMxArrayOnGPU(y);
    
    mxGPUDestroyGPUArray(Aval);
    mxGPUDestroyGPUArray(Aptr);
    mxGPUDestroyGPUArray(Acol);
    mxGPUDestroyGPUArray(x);
    mxGPUDestroyGPUArray(y);

    return;
}

