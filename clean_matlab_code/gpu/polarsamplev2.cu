#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <algorithm>
#include <cusp/complex.h>
#include "polargrid.h"
#include "cuda_sample.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"


#define	SXY	    prhs[0]
#define	SGRID	prhs[1]
#define	GRID_DIM	prhs[2]
#define	KLUT     prhs[3]
#define	KLUTS     prhs[4]
#define	KR     prhs[5]
#define	KB     prhs[6]


texture<float, 1, hipReadModeElementType> texRef;

void error_handle(hipError_t status = hipErrorLaunchFailure);

void error_handle(hipError_t status){
  if(status != hipSuccess){
    hipError_t s= hipGetLastError();
    if(s != hipSuccess){
      printf("%s\n",hipGetErrorString(s));
      exit(1);
    }
  }
}

float cpu_kb_weight(float2 grid_pos, float2 point_pos,
		    float * kb_table,
		    int kb_table_size,
		    float kb_table_scale){
  float dist_x = fabs(grid_pos.x-point_pos.x)*kb_table_scale;
  float dist_y = fabs(grid_pos.y-point_pos.y)*kb_table_scale;
  int ix = (int)dist_x;
  float fx = dist_x-ix;
  int iy = (int)dist_y;
  float fy = dist_y-iy;

  if(ix+1 < kb_table_size && iy+1 < kb_table_size){
    return (kb_table[ix]*(1.0f-fx) + kb_table[ix+1]*(fx)) *
      (kb_table[iy]*(1.0f-fy) + kb_table[iy+1]*(fy));     
  }
  return 0.0f;
}


__device__ float kb_weight(float grid_pos, float point_pos,
				    int kb_table_size,
				    float kb_table_scale){
  float dist_x = fabs(grid_pos-point_pos)*kb_table_scale;

  int ix = (int)dist_x;
  float fx = dist_x-ix;
  if(ix+1 < kb_table_size){
  return (tex1Dfetch(texRef,ix)*(1.0f-fx) + tex1Dfetch(texRef,ix+1)*(fx));     
  }
  return 0.0f;
}


__device__ float kb_weight_a(float grid_pos, float point_pos,
				    int k_r,   float kbeta){
  float dist_x = fabs(grid_pos-point_pos);
 
  if(dist_x<k_r){
    dist_x*=2/k_r;
    dist_x*=dist_x;
  return     cyl_bessel_i0f(kbeta* sqrtf( 1-dist_x ));
  }
  return 0.0f;
}


__device__ float kb_weight(float2 grid_pos, float2 point_pos,
				    int kb_table_size,
				    float kb_table_scale){
  float dist_x = fabs(grid_pos.x-point_pos.x)*kb_table_scale;
  float dist_y = fabs(grid_pos.y-point_pos.y)*kb_table_scale;

  int ix = (int)dist_x;
  float fx = dist_x-ix;
  int iy = (int)dist_y;
  float fy = dist_y-iy;

  if(ix+1 < kb_table_size && iy+1 < kb_table_size){
  return (tex1Dfetch(texRef,ix)*(1.0f-fx) + tex1Dfetch(texRef,ix+1)*(fx)) * 
    (tex1Dfetch(texRef,iy)*(1.0f-fy) + tex1Dfetch(texRef,iy+1)*(fy));     
  }
  return 0.0f;
}


__global__ void cuda_sample_kernel(cusp::complex<float> * point_pos,
				   cusp::complex<float> * grid_value, 
				   int npoints, uint2 grid_size,
				   int kb_table_size,
				   float kb_table_scale,
				   float kernel_radius,
				   float kernel_beta,
				    cusp::complex<float> * sample_value){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if(i < npoints){
    cusp::complex<float> sv=0;
    cusp::complex<float> pp=point_pos[i];
    float sx=pp.real();
    float sy=pp.imag();

    for(int y = max(0.0f,ceil(sy-kernel_radius));y<= min(floor(sy+kernel_radius),grid_size.y-1.0f);y++){
      if(y < 0 || y > grid_size.y-1){
	continue;
      }
      float  kby=kb_weight(y,sy,kb_table_size,kb_table_scale);
      //float  kby=kb_weight_a(y,sy,kernel_radius,kernel_beta);
 
      for(int x = max(0.0f,ceil(sx-kernel_radius));x<= min(floor(sx+kernel_radius),grid_size.x-1.0f);x++){
	if(x < 0 || x > grid_size.x-1){
	  continue;
	}
	sv += grid_value[y*grid_size.x+x]*kby*kb_weight(x,sx,kb_table_size, kb_table_scale);
	//sv += grid_value[y*grid_size.x+x]*kby*kb_weight_a(x,sx,kernel_radius, kernel_beta);
	
      }
    }  
    sample_value[i] = sv;

  }
}

void cuda_sample(cusp::complex<float> * point_pos,
		 cusp::complex<float> * grid_value, int npoints, 
		 uint2 grid_size,
		 float * kb_table,
		 int kb_table_size,
		 float kb_table_scale,
		 float kernel_radius, float kernel_beta,		 
		 cusp::complex<float> * sample_value){
  hipMemset(sample_value,0,sizeof( cusp::complex<float>)*npoints);

  size_t offset;
  hipBindTexture(&offset,texRef, kb_table, sizeof(float)*kb_table_size);
  if(offset != 0){
    printf("Error: Texture offset different than zero. Table not allocated with hipMalloc!%d\n");
    return;
  }

  int block_size = BLOCKSIZE;
  int grid = (npoints+block_size-1)/block_size;
  clock_t t_i = clock();
    cuda_sample_kernel<<<grid,block_size>>>( point_pos,
					     grid_value, npoints, 
					     grid_size,
					     kb_table_size,
					     kb_table_scale,
					     kernel_radius,kernel_beta,
					     sample_value);
    hipDeviceSynchronize();
    

  clock_t t_e = clock();
  error_handle();
  //  printf("%d iter in %5.1f ms\n",iter,(t_e-t_i)*1000.0/CLOCKS_PER_SEC);
}


void mexFunction(int nlhs, mxArray * plhs[], int nrhs,const mxArray * prhs[]){
      /* Initialize the MathWorks GPU API. */
    mxInitGPU();

 mxGPUArray const *samples_pos;
 mxGPUArray const *grid_values;
 mxGPUArray const *kernel_lookup_table;
// OUTPUT
mxGPUArray  *samples_values;

//double *grid_dim = mxGetPr(GRID_DIM);
//int *grid_dim1 =(int *) mxGetPr(GRID_DIM);
 int *grid_dim0=( int *) (mxGetData(GRID_DIM));
//mwSize const *grid_dim =(mwSize const *) mxGetPr(GRID_DIM);
// mwSize *grid_dim =(mwSize *) *grid_dim1;

mwSize *grid_dim=(mwSize *)grid_dim0; 

float kernel_lookup_table_scale = mxGetScalar(KLUTS);
float  kernel_radius = mxGetScalar(KR);
float  kernel_beta = mxGetScalar(KB);


// 
samples_pos = mxGPUCreateFromMxArray(SXY);
grid_values = mxGPUCreateFromMxArray(SGRID);
 kernel_lookup_table=mxGPUCreateFromMxArray(KLUT); 

//mexPrintf("Grid Dimensions %d x %d\n",(grid_dim[0]),(grid_dim[1]));
//mexPrintf("Grid Dimensions %d x %d\n",(grid_dim1[0]),(grid_dim1[1]));
//mexPrintf("-\n");
// 

int npoints =  (int)(mxGPUGetNumberOfElements(samples_pos));
int kernel_lookup_table_size = (int)(mxGPUGetNumberOfElements(kernel_lookup_table));
// mwSize ndim= 1;
//mwSize const grid_dim2[]={npoints};
  
mwSize const *sample_dim= mxGPUGetDimensions(samples_pos);
mwSize ndim= mxGPUGetNumberOfDimensions(samples_pos);
//plhs[0]  =mxCreateNumericArray(ndim,grid_dim2,mxSINGLE_CLASS,mxCOMPLEX);

//mexErrMsgTxt("gpuArray 0");

samples_values= mxGPUCreateGPUArray(ndim,sample_dim,mxSINGLE_CLASS,mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);

//samples_values= mxGPUCreateGPUArray(ndim,grid_dim2,mxINT32_CLASS,mxREAL, MX_GPU_INITIALIZE_VALUES);
//samples_values= 0;

//POINTERS

cusp::complex<float> * d_samples_pos = (cusp::complex<float>  *)(const cusp::complex<float>  *)(mxGPUGetDataReadOnly(samples_pos));
cusp::complex<float> * d_grid_values = (cusp::complex<float>  *)(const cusp::complex<float>  *)(mxGPUGetDataReadOnly(grid_values));
float * d_kernel_lookup_table = (float *)(const float  *)(mxGPUGetDataReadOnly(kernel_lookup_table));

uint2 grid_size = {grid_dim[0],grid_dim[1]};


// OUTPUT
cusp::complex<float> * d_samples_values = ( cusp::complex<float> *)(mxGPUGetData(samples_values));

 cuda_sample( d_samples_pos,
	       d_grid_values, npoints, 
	       grid_size, 
 	       d_kernel_lookup_table,
	       kernel_lookup_table_size,
	       kernel_lookup_table_scale,
	      kernel_radius, kernel_beta,
	       d_samples_values);  

  // GET OUTPUT
  plhs[0] = mxGPUCreateMxArrayOnGPU(samples_values);



 mxGPUDestroyGPUArray( samples_pos);
 mxGPUDestroyGPUArray( grid_values);
 mxGPUDestroyGPUArray( kernel_lookup_table);
 mxGPUDestroyGPUArray(samples_values);

}
