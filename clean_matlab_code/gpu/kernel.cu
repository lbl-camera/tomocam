#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <algorithm>
#include <vector>
#include <cusp/complex.h>
#include <hipcub/hipcub.hpp>
#include <thrust/reduce.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "hip/hip_runtime.h"
#include "polargrid.h"

texture<float, 1, hipReadModeElementType> texRef;

void error_handle(hipError_t status = hipErrorLaunchFailure);

void error_handle(hipError_t status){
    if(status != hipSuccess){
        hipError_t s= hipGetLastError();
        if(s != hipSuccess){
            //  printf("%s\n",hipGetErrorString(s));
            exit(1);
        }
    }
}


__host__ __device__ float weight(int2 grid_pos, float2 point_pos){
    return weight(make_float2(grid_pos.x,grid_pos.y),point_pos);
}

__host__ __device__ float weight(float2 grid_pos, float2 point_pos){
    if(fabs(grid_pos.x-point_pos.x) >= 3.0f ||
            fabs(grid_pos.y-point_pos.y) >= 3.0f){
        return 0;
    }
    return fabs(grid_pos.x-point_pos.x)+
            fabs(grid_pos.y-point_pos.y);
}

__device__ float kb_weight(float2 grid_pos, float2 point_pos,
        int kb_table_size,
        float kb_table_scale){
    float dist_x = fabsf(grid_pos.x-point_pos.x)*kb_table_scale;
    float dist_y = fabsf(grid_pos.y-point_pos.y)*kb_table_scale;
    
    int ix = (int)dist_x;
    float fx = dist_x-ix;
    int iy = (int)dist_y;
    float fy = dist_y-iy;
    
    if(ix+1 < kb_table_size && iy+1 < kb_table_size){
        return (tex1Dfetch<float>(texRef,ix)*(1.0f-fx) + tex1Dfetch<float>(texRef,ix+1)*(fx)) *
                (tex1Dfetch<float>(texRef,iy)*(1.0f-fy) + tex1Dfetch<float>(texRef,iy+1)*(fy));
    }
    return 0.0f;
}

__device__ float kb_weight(float grid_x, float grid_y, float point_pos_x,
        float point_pos_y,
        int kb_table_size,
        float kb_table_scale){
    float dist_x = fabsf(grid_x-point_pos_x)*kb_table_scale;
    float dist_y = fabsf(grid_y-point_pos_y)*kb_table_scale;
    
    int ix = (int)dist_x;
    float fx = dist_x-ix;
    int iy = (int)dist_y;
    float fy = dist_y-iy;
    
    if(ix+1 < kb_table_size && iy+1 < kb_table_size){
        return (tex1Dfetch<float>(texRef,ix)*(1.0f-fx) + tex1Dfetch<float>(texRef,ix+1)*(fx)) *
                (tex1Dfetch<float>(texRef,iy)*(1.0f-fy) + tex1Dfetch<float>(texRef,iy+1)*(fy));
    }
    return 0.0f;
}

__device__ float kb_weight(float2 grid_pos, float2 point_pos,
        int kb_table_size,
        float kb_table_scale,int tid){
    float dist_x = fabsf(grid_pos.x-point_pos.x)*kb_table_scale;
    float dist_y = fabsf(grid_pos.y-point_pos.y)*kb_table_scale;
    float ix = rintf(dist_x);
    float fx = dist_x-ix;
    float iy = rintf(dist_y);
    float fy = dist_y-iy;
    
    if(ix+1 < kb_table_size && iy+1 < kb_table_size){
        return (tex1Dfetch<float>(texRef,tid)*(1.0f-fx) + tex1Dfetch<float>(texRef,tid)*(fx)) *
                (tex1Dfetch<float>(texRef,tid)*(1.0f-fy) + tex1Dfetch<float>(texRef,tid)*(fy));
    }
    return 0.0f;
}


__global__ void sum_points(const cusp::complex<float> * point_value,
        const int *  binned_points,
        const float * binned_points_x,
        const float * binned_points_y,
        int points_per_bin, int idx,
        int x, int y, int kb_table_size, float kb_table_scale, cusp::complex<float> * grid_value)
{
    int tid = threadIdx.x;
    cusp::complex<float> sum_t[BLOCKSIZE];
   
// sum all points
        sum_t[tid] = 0;
        for(int j = tid;j<points_per_bin;j+=blockDim.x){
            sum_t[tid] += point_value[binned_points[idx+j]]*kb_weight(make_float2(x,y),
                    make_float2(binned_points_x[idx+j],
                    binned_points_y[idx+j]),
                    kb_table_size,kb_table_scale);
        }
        for(unsigned int j=1; j < blockDim.x; j *= 2) {
            // modulo arithmetic is slow!
            if ((tid & (2*j-1)) == 0) {
                sum_t[tid] += sum_t[tid + j];
            }
            __syncthreads();
        }
        if(tid == 0){
            grid_value[0]=sum_t[0];
            return;
        }
}


//
// __device__ float kb_weight(float2 grid_pos, float2 point_pos,
//         float * kb_table, int kb_table_size,
//         float kb_table_scale){
//     float dist_x = fabsf(grid_pos.x-point_pos.x)*kb_table_scale;
//     float dist_y = fabsf(grid_pos.y-point_pos.y)*kb_table_scale;
//     int ix = (int)dist_x;
//     float fx = dist_x-rintf(dist_x);
//     int iy = (int)dist_y;
//     float fy = dist_y-rintf(dist_y);
//
//     if(ix+1 < kb_table_size && iy+1 < kb_table_size){
//         return (kb_table[ix]*(1.0f-fx) + kb_table[ix+1]*(fx)) *
//                 (kb_table[iy]*(1.0f-fy) + kb_table[iy+1]*(fy));
//     }
//     return 0.0f;
// }
/*
__global__ void grid_points_cuda_mex_interleaved_kernel(const float * point_x,
        const float * point_y,
        const cusp::complex<float> * point_value,
        int npoints,  uint2 grid_size,
        const int *  points_per_bin,
        const int * bin_dimension_x,
        const int * bin_dimension_y,
        const int *  binned_points,
        const int * binned_points_idx,
        const int * bin_location,
        const float * binned_points_x,
        const float * binned_points_y,
        int nbins,
        int kb_table_size,
        float kb_table_scale,
        cusp::complex<float> * grid_value){
    
    
    // Specialize BlockReduce for a 1D block of 128 threads on type cusp::complex<float>
    //typedef hipcub::BlockReduce<cusp::complex<float>, 128> BlockReduce;
    // Allocate shared memory for BlockReduce
    //__shared__ typename BlockReduce::TempStorage temp_storage;
    //int aggregate = BlockReduce(temp_storage).Sum(thread_data);
    
    int i = blockIdx.x;
    int tid = threadIdx.x;
    uint2 corner;
    corner.x = bin_location[i]%grid_size.x;
    corner.y = bin_location[i]/grid_size.x;
    int idx = binned_points_idx[i];
    __shared__ float point_pos_cache_x[SHARED_SIZE];
    __shared__ float point_pos_cache_y[SHARED_SIZE];
    __shared__ cusp::complex<float> point_value_cache[SHARED_SIZE];
   
    __shared__ cusp::complex<float> sum_t[BLOCKSIZE];
    
    // small bin or large no of samples
    if(bin_dimension_x[i]*bin_dimension_y[i] < 64 || points_per_bin[i] > SHARED_SIZE){
        sum_t[tid] = 0;
//    loop through grid
        for(int y = corner.y;y<corner.y+bin_dimension_x[i];y+=1){
            for(int x = corner.x;x<corner.x+bin_dimension_y[i];x+=1){
                sum_t[tid] = 0;
                // loop through points
                for(int j = tid;j<points_per_bin[i];j+=blockDim.x){
                    sum_t[tid] += point_value[binned_points[idx+j]]*kb_weight(make_float2(x,y),
                            make_float2(binned_points_x[idx+j],
                            binned_points_y[idx+j]),
                            kb_table_size,kb_table_scale);
                }
                // Do a reduce in shared memory //
                for(unsigned int j=1; j < blockDim.x; j *= 2) {
                    // modulo arithmetic is slow!
                    if ((tid & (2*j-1)) == 0) {
                        sum_t[tid] += sum_t[tid + j];
                    }
                    __syncthreads();
                }
                if(tid == 0){
                    grid_value[y*grid_size.x+x] = sum_t[0];
                }
            }
        }
        // large dimensions
    }else if(bin_dimension_x[i]*bin_dimension_y[i] >BLOCKSIZE/2-1) {
        // Lets try to load all points to shared memory /
        const int ppb = points_per_bin[i];
        for(int j = tid;j<ppb;j+= blockDim.x){
            const int point = binned_points[idx+j];
            point_value_cache[j] = point_value[point];
            point_pos_cache_x[j] = binned_points_x[idx+j];
            point_pos_cache_y[j] = binned_points_y[idx+j];
        }
        __syncthreads();
        const uint2 dims = {bin_dimension_x[i],bin_dimension_y[i]};
        // loop through dimensions
        for(int k = tid;k<dims.x*dims.y;k+=blockDim.x){
            const int x = (k%(dims.x))+corner.x;
            const int y = (k/dims.x)+corner.y;
            cusp::complex<float> my_sum = 0;
            for(int j = 0;j<ppb;j++){ //loop through all the points
                float w=                      kb_weight(x,y,point_pos_cache_x[j],point_pos_cache_y[j],kb_table_size,kb_table_scale);
                my_sum += point_value_cache[j]*w;
            }
            grid_value[y*grid_size.x+x] = my_sum;
        }
    }else{ //small dimension and few points
        // Lets try to load things to shared memory /
        const int ppb = points_per_bin[i];
        for(int j = tid;j<ppb;j+= blockDim.x){
            const int point = binned_points[idx+j];
            point_value_cache[j] = point_value[point];
            point_pos_cache_x[j] = binned_points_x[idx+j];
            point_pos_cache_y[j] = binned_points_y[idx+j];
        }
        __syncthreads();
        const uint2 dims = {bin_dimension_x[i],bin_dimension_y[i]};
        int b = 4;
        for(int k = tid/b;k<dims.x*dims.y;k+=blockDim.x/b){
            const int x = (k%(dims.x))+corner.x;
            const int y = (k/dims.x)+corner.y;
            sum_t[tid] = 0;
            //sum_i[tid] = 0;
            for(int j = (tid&(b-1));j<ppb;j+=b){
                float w= kb_weight(x,y,point_pos_cache_x[j],point_pos_cache_y[j],kb_table_size,kb_table_scale);
                sum_t[tid] += point_value_cache[j]*w;
            }
            // Do a reduce in shared memory 
            for(unsigned int j=1; j < b; j = (j << 1)) {
                // modulo arithmetic is slow!
                if ((tid & ((j<<1)-1)) == 0) {
                    sum_t[tid] += sum_t[tid + j];
                    
                }
                __syncthreads();
            }
            if((tid&(b-1)) == 0){
                grid_value[y*grid_size.x+x] = sum_t[tid];
                
            }
        }
    }
}
*/
//------------------------------
//point_value[binned_points[idx+j]]*kb_weight(make_float2(x,y),
//                           make_float2(binned_points_x[idx+j],
//                          binned_points_y[idx+j]),
//                          kb_table_size,kb_table_scale);

//call=
//transform_4in1out( binned_points_x,binned_points_y,binned_points,point_value,make_float2(x,y), kb_table_size,kb_table_scale);
 
/*
template <typename IN, typename OUT>
         struct KBMUL
        {
            float xs;
            float ys;
            int kb_table_size;
            float kb_table_scale;
                        
            KBMUL(float _xs, float _ys){
                xs = _xs;
                ys= _ys;
                kb_table_size=_kb_table_size;
                kb_table_scale=_kb_table_scale;
            }
                                
            template <typename Tuple>
                    __host__ __device__
                    OUT operator()(Tuple x)
            {
//                 OUT out;
                 IN point_value= thrust::get<0>(x);
                 float binned_points_x= thrust::get<1>(x);
                 float binned_points_y= thrust::get<2>(x);
                 
                   OUT  ret = point_value[binned_points]*kb_weight(make_float2(xs,ys),
                            make_float2(binned_points_x,binned_points_y),
                            kb_table_size,kb_table_scale);
                   return ret*ret;
            }
};
*/

//------------------------
/* 
 *                    sum_t[tid] += point_value[binned_points[idx+j]]*kb_weight(make_float2(x,y),
                            make_float2(binned_points_x[idx+j],
                            binned_points_y[idx+j]),
                            kb_table_size,kb_table_scale);

 * template <typename T1,typename T2>

 * struct AbsSubtract2 : public thrust::unary_function<T1,T2>
{
  __host__ __device__
  T2 operator()(T1 x)
  {  
    T2 ret = abs(abs(thrust::get<0>(x))-(thrust::get<1>(x)));
    return ret*ret;
  }
};
              */
//=========================
/*            
template<typename IN,typename OUT>
  void transform_4in_1out(float * BINNED_POINTS_X, float * BINNED_POINTS_Y, int BINNED_POINTS, IN
        * POINT_VALUE, float2 * POSITIONS, 
        int kb_table_size, float kb_table_scale, float * KBTABLE, OUT * derr, int N){
    thrust::device_ptr<float> d_binned_points_x(BINNED_POINTS_X);
    thrust::device_ptr<float> d_binned_points_y(BINNED_POINTS_Y);
    thrust::device_ptr<int> d_binned_points(BINNED_POINTS);
    thrust::device_ptr<float> d_value(POINT_VALUE);
    thrust::device_ptr<float> d_positions(POSITIONS);

    
}
//          transform_3in_2out(d_G,d_DG,d_a,  (float ) tau, &der,&d2er,n);
//          transform_3in_1out(d_G,d_DG,d_a,  (float ) tau, &der,n);

        template<typename IN,typename OUT>
//                void transform_3in_2out(IN * G, IN * dG, float * F, float tau, OUT * derr, OUT * d2err, int N){
                void transform_3in_2out(IN * G, IN * dG, float * F, float tau, OUT * derr, OUT * d2err, int N){
            thrust::device_ptr<IN> d_G(G);
            thrust::device_ptr<IN> d_dG(dG);
            thrust::device_ptr<float> d_F(F);
            thrust::tuple<OUT,OUT> init;
            thrust::tuple<OUT,OUT> out = thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(d_G, d_dG, d_F)),
                    thrust::make_zip_iterator(thrust::make_tuple(d_G, d_dG, d_F))+N,
                    DIR<IN,OUT>(tau),
                    init,
                    TUPLE_PLUS<thrust::tuple<OUT,OUT> >());
            *derr = thrust::get<0>(out)*2;
            *d2err = thrust::get<1>(out)*2;
        }
*/
        
//--------------------        
__global__ void sum_points(const cusp::complex<float> * point_value,
        const int *  binned_points,
        const float * binned_points_x,
        const float * binned_points_y,
        int points_per_bin,
        int idx,
        int x, int y, int kb_table_size, float kb_table_scale, cusp::complex<float> * grid_value)
{
    int tid = threadIdx.x;
    cusp::complex<float> sum_t[BLOCKSIZE];
    
// sum all points
        sum_t[tid] = 0;
        for(int j = tid;j<points_per_bin;j+=blockDim.x){
            sum_t[tid] += point_value[binned_points[idx+j]]*kb_weight(make_float2(x,y),
                    make_float2(binned_points_x[idx+j],
                    binned_points_y[idx+j]),
                    kb_table_size,kb_table_scale);
        }
        for(unsigned int j=1; j < blockDim.x; j *= 2) {
            // modulo arithmetic is slow!
            if ((tid & (2*j-1)) == 0) {
                sum_t[tid] += sum_t[tid + j];
            }
            __syncthreads();
        }
        if(tid == 0){
            grid_value[0]=sum_t[0];
            return;
        }

//                    return ret;
//                    return sum_t[0];

    }




//--------------------------------
 __global__ void grid_points_cuda_mex_interleaved_kernel1(const float * point_x,
                const float * point_y,
                const cusp::complex<float> * point_value,
                int npoints,  uint2 grid_size,
                const int *  points_per_bin,
                const int * bin_dimension_x,
                const int * bin_dimension_y,
                const int *  binned_points,
                const int * binned_points_idx,
                const int * bin_location,
                const float * binned_points_x,
                const float * binned_points_y,
                int nbins,
                int kb_table_size,
                float kb_table_scale,
                cusp::complex<float> * grid_value){
            
            __shared__ float point_pos_cache_x[SHARED_SIZE];
            __shared__ float point_pos_cache_y[SHARED_SIZE];
            __shared__ cusp::complex<float> point_value_cache[SHARED_SIZE];
            
            __shared__ cusp::complex<float> sum_t[BLOCKSIZE];
            
            
            int i = blockIdx.x;
            int tid = threadIdx.x;
            uint2 corner;
            corner.x = bin_location[i]%grid_size.x;
            corner.y = bin_location[i]/grid_size.x;
            int idx = binned_points_idx[i];
           const int ppb = points_per_bin[i];
                         
            // small bin or large no of samples
            if(bin_dimension_x[i]*bin_dimension_y[i] < 64 || points_per_bin[i] > SHARED_SIZE){
//    loop through grid
                for(int y = corner.y;y<corner.y+bin_dimension_x[i];y+=1){
                    for(int x = corner.x;x<corner.x+bin_dimension_y[i];x+=1){
//                sum_points<<<1,BLOCKSIZE>>> (point_value,binned_points,binned_points_x,binned_points_y,idx,points_per_bin[idx],x,y,kb_table_size,kb_table_scale, value);
//             cusp::complex<float> value[1];
//grid_value[y*grid_size.x+x]=0;
 sum_points<<<1,BLOCKSIZE>>> (point_value,binned_points,binned_points_x,binned_points_y,idx,ppb,x,y,kb_table_size,kb_table_scale,grid_value+y*grid_size.x+x);
             
            /*
                sum_t[tid] = 0;
                        // Specialize BlockReduce for a 1D block of 128 threads on type cusp::complex<float>
                        //typedef hipcub::BlockReduce<cusp::complex<float>, 128> BlockReduce;
                        // Allocate shared memory for BlockReduce
                        //__shared__ typename BlockReduce::TempStorage temp_storage;
                        
                        //  grid_value[y*grid_size.x+x]= BlockReduce(temp_storage).Sum(thread_data);
                        //                for(int item=0; item<ITEMS_PER_THREAD; ++item)
//    data[item] = unaryOp(data[item]);
                        
                        sum_t[tid] = 0;
                        // loop through points
                        for(int j = tid;j<ppb;j+=blockDim.x){
                            sum_t[tid] += point_value[binned_points[idx+j]]*kb_weight(make_float2(x,y),
                                    make_float2(binned_points_x[idx+j],
                                    binned_points_y[idx+j]),
                                    kb_table_size,kb_table_scale);
                        }
                        // Do a reduce in shared memory 
                        for(unsigned int j=1; j < blockDim.x; j *= 2) {
                            // modulo arithmetic is slow!
                            if ((tid & (2*j-1)) == 0) {
                                sum_t[tid] += sum_t[tid + j];
                            }
                            __syncthreads();
                        }
                        if(tid == 0){
                            grid_value[y*grid_size.x+x] = sum_t[0];
                        }
                        */
                         
                    }
                }
                // large dimensions
            }else if(bin_dimension_x[i]*bin_dimension_y[i] >BLOCKSIZE/2-1) {
                /* Lets try to load all points to shared memory */
                const int ppb = points_per_bin[i];
                for(int j = tid;j<ppb;j+= blockDim.x){
                    const int point = binned_points[idx+j];
                    point_value_cache[j] = point_value[point];
                    point_pos_cache_x[j] = binned_points_x[idx+j];
                    point_pos_cache_y[j] = binned_points_y[idx+j];
                }
                __syncthreads();
                const uint2 dims = {bin_dimension_x[i],bin_dimension_y[i]};
                // loop through dimensions
                for(int k = tid;k<dims.x*dims.y;k+=blockDim.x){
                    const int x = (k%(dims.x))+corner.x;
                    const int y = (k/dims.x)+corner.y;
                    cusp::complex<float> my_sum = 0;
                    for(int j = 0;j<ppb;j++){ //loop through all the points
                        float w=                      kb_weight(x,y,point_pos_cache_x[j],point_pos_cache_y[j],kb_table_size,kb_table_scale);
                        my_sum += point_value_cache[j]*w;
                    }
                    grid_value[y*grid_size.x+x] = my_sum;
                }
            }else{ //small dimension and few points
                /* Lets try to load things to shared memory */
                const int ppb = points_per_bin[i];
                for(int j = tid;j<ppb;j+= blockDim.x){
                    const int point = binned_points[idx+j];
                    point_value_cache[j] = point_value[point];
                    point_pos_cache_x[j] = binned_points_x[idx+j];
                    point_pos_cache_y[j] = binned_points_y[idx+j];
                }
                __syncthreads();
                const uint2 dims = {bin_dimension_x[i],bin_dimension_y[i]};
                int b = 4;
                for(int k = tid/b;k<dims.x*dims.y;k+=blockDim.x/b){
                    const int x = (k%(dims.x))+corner.x;
                    const int y = (k/dims.x)+corner.y;
                    sum_t[tid] = 0;
                    //sum_i[tid] = 0;
                    for(int j = (tid&(b-1));j<ppb;j+=b){
                        float w= kb_weight(x,y,point_pos_cache_x[j],point_pos_cache_y[j],kb_table_size,kb_table_scale);
                        sum_t[tid] += point_value_cache[j]*w;
                    }
                    /* Do a reduce in shared memory */
                    for(unsigned int j=1; j < b; j = (j << 1)) {
                        // modulo arithmetic is slow!
                        if ((tid & ((j<<1)-1)) == 0) {
                            sum_t[tid] += sum_t[tid + j];
                            
                        }
                        __syncthreads();
                    }
                    if((tid&(b-1)) == 0){
                        grid_value[y*grid_size.x+x] = sum_t[tid];
                        
                    }
                }
            }
        }
        
        
        void grid_points_cuda_interleaved_mex(const float * point_pos_x, const float * point_pos_y,
                const cusp::complex<float> * point_value, int npoints,
                uint2 grid_size, const int * points_per_bin, const int * bin_dimension_x,
                const int * bin_dimension_y,
                const int * binned_points, const int * binned_points_idx, const int * bin_location,
                const float * binned_points_x, const float * binned_points_y,
                int nbins,
                const float * kb_table,
                int kb_table_size,
                float kb_table_scale,
                cusp::complex<float> * grid_value){
            hipMemset(grid_value,0,sizeof(float2)*grid_size.x*grid_size.y);
            
            size_t offset;
            hipBindTexture(&offset,texRef, kb_table, sizeof(float)*kb_table_size);
            if(offset != 0){
                //   printf("Error: Texture offset different than zero. Table not allocated with hipMalloc!%d\n");
                return;
            }
            
            int grid = nbins;
            int block_size = BLOCKSIZE;
            clock_t t_i = clock();
            grid_points_cuda_mex_interleaved_kernel1<<<grid,block_size>>>( point_pos_x, point_pos_y,
                    point_value, npoints, grid_size, points_per_bin,
                    bin_dimension_x, bin_dimension_y, binned_points,
                    binned_points_idx, bin_location,
                    binned_points_x, binned_points_y,nbins,
                    kb_table_size,
                    kb_table_scale,
                    grid_value);
            hipDeviceSynchronize();
            
            clock_t t_e = clock();
            error_handle();
            //  printf("%d iter in %5.1f ms\n",iter,(t_e-t_i)*1000.0/CLOCKS_PER_SEC);
        }
        
        void compare_to_gold(float * gridded, float * gold_gridded, uint2 grid_size){
            for(int i =0;i<grid_size.x*grid_size.y;i++){
                if(fabs(gridded[i]-gold_gridded[i])/gridded[i] > 1e-5 &&
                        fabs(gridded[i]-gold_gridded[i]) > 1e-7){
//      printf("cuda[%d] = %e gold[%d] = %e\n",i,gridded[i],i,gold_gridded[i]);
                    exit(1);
                }
            }
        }
        
        
        
