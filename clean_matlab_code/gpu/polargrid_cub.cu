#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <algorithm>
#include <vector>
#include <cusp/complex.h>
#include <cusp/blas.h>
#include <hipcub/hipcub.hpp>
#include <thrust/reduce.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "hip/hip_runtime.h"
#include "polargrid.h"
//#include "stdafx.h"
#include "hip/hip_runtime_api.h"

texture<int,1> tex_x_int;
texture<float,1> tex_x_float;
texture<float,1> tex_x_float1;


__inline__ __device__ cusp::complex<float> fetch_x(const int& i, const cusp::complex<float> * x)
{
return cusp::complex<float>(tex1Dfetch(tex_x_float, i*2),tex1Dfetch(tex_x_float, i*2+1));
}

__inline__ __device__ float fetch_x(const int& i,const float * x)
{
return tex1Dfetch(tex_x_float1, i);
}

__inline__ __device__ cusp::complex<float> fetch_xc(const int& i, const cusp::complex<float> * x)
{
return cusp::complex<float>(tex1Dfetch(tex_x_float, i*2),tex1Dfetch(tex_x_float, i*2+1));
}

__inline__ __device__ cusp::complex<float> shflb( const cusp::complex<float>  x, int& i)
{
  //  return cusp::complex<float>(__shfl(float2(x),i));
  //  return cusp::complex<float>(__shfl(float(x),i*2),(__shfl(float(x),i*2+1)));
  return cusp::complex<float>(__shfl(x.x,i),(__shfl(x.y,i)));
}

__inline__ __device__ float shflb( const float  x, int& i)
{
  return float(__shfl(x,i));
}

__inline__ __device__ void atomicAdd( cusp::complex<float> * x,  cusp::complex<float>  m)
{
  //  cusp::complex<float> m;
     atomicAdd(&(x[0]).x,m.x);
     atomicAdd(&(x[0]).y,m.y);
      //      return m;
}



void error_handle(hipError_t status = hipErrorLaunchFailure);

void error_handle(hipError_t status){
    if(status != hipSuccess){
        hipError_t s= hipGetLastError();
        if(s != hipSuccess){
            //  printf("%s\n",hipGetErrorString(s));
            exit(1);
        }
    }
}



__device__ float kb_weight(float2 grid_pos, float2 point_pos,
        int kb_table_size, float kb_table_scale,                const float * kb_table){
    float dist_x = fabsf(grid_pos.x-point_pos.x)*kb_table_scale;
    float dist_y = fabsf(grid_pos.y-point_pos.y)*kb_table_scale;
    //float w=tex1D<float>(texRef,0.0f);

    //    return w;//tex1D<float>(texRef,dist_y);// *tex1D<float>(texRef,dist_y);

//      return 1.0f;
    
    int ix = (int)dist_x;
    float fx = dist_x-ix;
    int iy = (int)dist_y;
    float fy = dist_y-iy;
    
    if(ix+1 < kb_table_size && iy+1 < kb_table_size){
      //  return (tex1Dfetch<float>(texRef,ix)*(1.0f-fx) + tex1Dfetch<float>(texRef,ix+1)*(fx)) *
      //           (tex1Dfetch<float>(texRef,iy)*(1.0f-fy) + tex1Dfetch<float>(texRef,iy+1)*(fy));

return	(fetch_x(ix,kb_table)*(1.0f-fx)+ fetch_x(ix+1,kb_table)*(fx))* 
	(fetch_x(iy,kb_table)*(1.0f-fy)+ fetch_x(iy+1,kb_table)*(fy)); 
    }
    return 0.0f;
    /*    */
}

__device__ float kb_weight(float grid_x, float grid_y, float point_pos_x,
        float point_pos_y,
        int kb_table_size,
    float kb_table_scale, const float * kb_table){
    float dist_x = fabsf(grid_x-point_pos_x)*kb_table_scale;
    float dist_y = fabsf(grid_y-point_pos_y)*kb_table_scale;
    //    return tex1D<float>(texRef,dist_x) *tex1D<float>(texRef,dist_y);


     int ix = (int)dist_x;
    float fx = dist_x-ix;
    int iy = (int)dist_y;
    float fy = dist_y-iy;
    
    if(ix+1 < kb_table_size && iy+1 < kb_table_size){
return	(fetch_x(ix,kb_table)*(1.0f-fx)+ fetch_x(ix+1,kb_table)*(fx))* 
	(fetch_x(iy,kb_table)*(1.0f-fy)+ fetch_x(iy+1,kb_table)*(fy)); 
/*
        return (tex1Dfetch<float>(texRef,ix)*(1.0f-fx) + tex1Dfetch<float>(texRef,ix+1)*(fx)) *
                (tex1Dfetch<float>(texRef,iy)*(1.0f-fy) + tex1Dfetch<float>(texRef,iy+1)*(fy));
*/

    }
    return 0.0f;

}

__device__ float kb_weight(float2 grid_pos, float2 point_pos,
        int kb_table_size,
			   float kb_table_scale,                const float * kb_table,int tid){
    float dist_x = fabsf(grid_pos.x-point_pos.x)*kb_table_scale;
    float dist_y = fabsf(grid_pos.y-point_pos.y)*kb_table_scale;
    //  return 0.0f;
    //    return tex1D<float>(texRef,dist_x) *tex1D<float>(texRef,dist_y);


    //    return tex1D<float>(texRef,dist_x) *tex1D<float>(texRef,dist_y);

  
    float ix = rintf(dist_x);
    float fx = dist_x-ix;
    float iy = rintf(dist_y);
    float fy = dist_y-iy;
    
    if(ix+1 < kb_table_size && iy+1 < kb_table_size){

return	(fetch_x(ix,kb_table)*(1.0f-fx)+ fetch_x(ix+1,kb_table)*(fx))* 
	(fetch_x(iy,kb_table)*(1.0f-fy)+ fetch_x(iy+1,kb_table)*(fy)); 

/*
        return (tex1Dfetch<float>(texRef,tid)*(1.0f-fx) + tex1Dfetch<float>(texRef,tid)*(fx)) *
                (tex1Dfetch<float>(texRef,tid)*(1.0f-fy) + tex1Dfetch<float>(texRef,tid)*(fy));
*/
    }
    return 0.0f;
  

}


//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
__global__ void sum_points(        const cusp::complex<float> * point_value,
        int npoints,  uint2 grid_size,
        const int *  points_per_bin,
        const int * bin_dimension_x,
        const int * bin_dimension_y,
        const int *  binned_points,
        const int * binned_points_idx,
        const int * bin_location,
        const float * binned_points_x,
        const float * binned_points_y,
        const int nbins,
        const int kb_table_size,
        const float kb_table_scale,                const float * kb_table, hipTextureObject_t texRef,
			   cusp::complex<float> * grid_value,int pbid){
  __shared__ cusp::complex<float> value;
  
  __shared__ cusp::complex<float> sum_t[BLOCKSIZE];
    
  // Specialize BlockReduce for a 1D block of 128 threads on type cusp::complex<float>
  
  //    int i = blockIdx.x;

  int i = pbid;
  int tid = threadIdx.x;
  
  int jj = blockIdx.x;
  
  uint2 corner;
  corner.x = bin_location[i]%grid_size.x;
  corner.y = bin_location[i]/grid_size.x;
  const int idx = binned_points_idx[i];
  const int ppb = points_per_bin[i];
  //    cusp::complex<float> * value;
  const int  bd=BLOCKSIZE;
  //	const int  bd=blockDim.x;
  //const uint2 dims = {bin_dimension_x[i],bin_dimension_y[i]};
            
  
  int bdx=bin_dimension_x[i];
  //    loop through grid
  for(int yi = corner.y;yi<corner.y+bin_dimension_x[i];yi+=1){
    int y=(yi-corner.y+jj)%bdx+corner.y; //shift so that there is no overlap
    
    //	  int y=yi;
    for(int x = corner.x;x<corner.x+bin_dimension_y[i];x+=1){
      
      sum_t[tid] = 0;
      
      for(int j = tid+jj*bd ;j<ppb;j+=bd*gridDim.x){
	sum_t[tid] += point_value[binned_points[idx+j]]*
	  kb_weight(make_float2(x,y),
		    make_float2(binned_points_x[idx+j],binned_points_y[idx+j]),
		    kb_table_size,kb_table_scale, kb_table,texRef);
      }
      __syncthreads();
      
      for(unsigned int j=1; j < bd; j *= 2) {
	// modulo arithmetic is slow!
	if ((tid & (2*j-1)) == 0) { sum_t[tid] += sum_t[tid + j];  }
	__syncthreads();
      }
      
      hipDeviceSynchronize();
      
      if(tid == 0){
	//	  grid_value[y*grid_size.x+x]+=(cusp::complex<float>) sum_t[0]; 
	atomicAdd(&(grid_value[y*grid_size.x+x]),(sum_t[0]));
      }
    }
  }
}



//------------------------------
__global__ void grid_points_cuda_mex_interleaved_kernel(
        const cusp::complex<float> * point_value,
        int npoints,  uint2 grid_size,
        const int *  points_per_bin,
        const int * bin_dimension_x,
        const int * bin_dimension_y,
        const int *  binned_points,
        const int * binned_points_idx,
        const int * bin_location,
        const float * binned_points_x,
        const float * binned_points_y,
        const int nbins,
        const int kb_table_size,
							const float kb_table_scale,                 const float * kb_table,//hipTextureObject_t texRef,
	size_t offset,size_t kernel_tex,
	cusp::complex<float> * grid_value){
  __shared__ cusp::complex<float> value;
      
    // Specialize BlockReduce for a 1D block of 128 threads on type cusp::complex<float>
    
    int i = blockIdx.x;
        if(points_per_bin[i]==0){return;}

    int tid = threadIdx.x;
    uint2 corner;
    corner.x = bin_location[i]%grid_size.x;
    corner.y = bin_location[i]/grid_size.x;

    const int idx = binned_points_idx[i];
    const int ppb = points_per_bin[i];
    //    cusp::complex<float> * value;
    const int  bd=BLOCKSIZE;
    //	const int  bd=blockDim.x;
    const uint2 dims = {bin_dimension_x[i],bin_dimension_y[i]};
            


    // small bin or large no of samples
    if(bin_dimension_x[i]*bin_dimension_y[i] < 64 || points_per_bin[i] > SHARED_SIZE){
      //      return;


	  /*
	  sum_points<<<3,BLOCKSIZE>>>( point_x, point_y,
                    point_value, npoints, grid_size, points_per_bin,
                    bin_dimension_x, bin_dimension_y, binned_points,
                    binned_points_idx, bin_location,
                    binned_points_x, binned_points_y,nbins,
                    kb_table_size,
		    kb_table_scale, kb_table, 
		  grid_value, blockIdx.x);
	  */
	  
          __shared__ cusp::complex<float> sum_t[bd];

//    loop through grid
        for(int y = corner.y;y<corner.y+bin_dimension_x[i];y+=1){
            for(int x = corner.x;x<corner.x+bin_dimension_y[i];x+=1){
        sum_t[tid] = 0;

        for(int j = tid;j<ppb;j+=bd){
	    sum_t[tid] +=   fetch_x(binned_points[idx+j],point_value)* 
		    //sum_t[tid] += point_value[binned_points[idx+j]]*
          kb_weight(make_float2(x,y),
                    make_float2(binned_points_x[idx+j],binned_points_y[idx+j]),
			      kb_table_size,kb_table_scale, kb_table);

        }
	//          __syncthreads();

        for(unsigned int j=1; j < bd; j *= 2) {
            // modulo arithmetic is slow!
            if ((tid & (2*j-1)) == 0) { sum_t[tid] += sum_t[tid + j];  }
	                __syncthreads();
        }

        if(tid == 0){
	  grid_value[y*grid_size.x+x]=sum_t[0]; 
	  //	atomicAdd(&(grid_value[y*grid_size.x+x]).x,(sum_t[0]).x);
	  //atomicAdd(&(grid_value[y*grid_size.x+x]).y,(sum_t[0]).y);

}
                
            }
        }        // large dimensions
    }else{ //small dimension and few points


  int bblock;
    if(bin_dimension_x[i]*bin_dimension_y[i] >BLOCKSIZE/2-1)
      bblock=1;
    else {bblock=4; };

    __shared__ float point_pos_cache_x[SHARED_SIZE];
    __shared__ float point_pos_cache_y[SHARED_SIZE];
    __shared__ cusp::complex<float> point_value_cache[SHARED_SIZE];  
    __shared__ cusp::complex<float> sum_t[BLOCKSIZE];
    //  

    //    bblock=4;

        // Lets try to load things to shared memory /
        for(int j = tid;j<ppb;j+= blockDim.x){
	  // const int point = binned_points[idx+j];
	      point_value_cache[j] =fetch_x(binned_points[idx+j],point_value); 
	      //              point_value_cache[j] = point_value[point];
            point_pos_cache_x[j] = binned_points_x[idx+j];
            point_pos_cache_y[j] = binned_points_y[idx+j];
        }
        __syncthreads();
        int b = bblock;// each thread takes care of 

        for(int k = tid/b;k<dims.x*dims.y;k+=blockDim.x/b){
            const int x = (k%(dims.x))+corner.x;
            const int y = (k/dims.x)+corner.y;
            sum_t[tid] = 0;
            //sum_i[tid] = 0;
            for(int j = (tid&(b-1));j<ppb;j+=b){
	      //                float w= kb_weight(x,y,point_pos_cache_x[j],point_pos_cache_y[j],kb_table_size,kb_table_scale, kb_table,texRef);

                float w=kb_weight(make_float2(x,y),make_float2(point_pos_cache_x[j],point_pos_cache_y[j]),
			       kb_table_size,kb_table_scale, kb_table);


                sum_t[tid] += point_value_cache[j]*w;
            }
            // Do a reduce in shared memory 
                __syncthreads();

            for(unsigned int j=1; j < b; j = (j << 1)) {
                // modulo arithmetic is slow!
                if ((tid & ((j<<1)-1)) == 0) {
                    sum_t[tid] += sum_t[tid + j];
                }
            }
            if((tid&(b-1)) == 0){
                grid_value[y*grid_size.x+x] = sum_t[tid];
                
            }
        }
    }
}

//--------------------------------
        void grid_points_cuda_interleaved_mex(const cusp::complex<float> * point_value, int npoints,
                uint2 grid_size, const int * points_per_bin, const int * bin_dimension_x,
                const int * bin_dimension_y,
                const int * binned_points, const int * binned_points_idx, const int * bin_location,
                const float * binned_points_x, const float * binned_points_y,
                int nbins,
                const float * kb_table,
                const int kb_table_size, const float kb_table_scale, cusp::complex<float> * grid_value){
            hipMemset(grid_value,0,sizeof(float2)*grid_size.x*grid_size.y);


           int grid = nbins;
            int block_size = BLOCKSIZE;
            clock_t t_i = clock();


  /*

  // create texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  if(0){
 hipChannelFormatDesc channelDesc=hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat );
 hipArray * cuArray;
 hipMallocArray(&cuArray, &channelDesc, kb_table_size, 1);

 hipMemcpyToArray ( cuArray,  0,0,kb_table,kb_table_size*sizeof(float) , hipMemcpyDeviceToDevice);

      resDesc.resType = hipResourceTypeArray;
     resDesc.res.array.array = cuArray;

  }
  else{
   resDesc.resType = hipResourceTypeLinear;
   resDesc.res.linear.devPtr = (void *)kb_table;

    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
 resDesc.res.linear.desc.x = 32; // bits per channel
 resDesc.res.linear.sizeInBytes =  kb_table_size*sizeof(float);
  }


  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.readMode = hipReadModeElementType;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.normalizedCoords = 1;

  // create texture object: we only have to do this once!
  hipTextureObject_t texRef=0;
  hipCreateTextureObject(&texRef, &resDesc, &texDesc, NULL);

//--------------------------------------------
//  if(1){
  // create texture object
 hipResourceDesc res1Desc;
 memset(&res1Desc, 0, sizeof(res1Desc));
   res1Desc.resType = hipResourceTypeLinear;
   res1Desc.res.linear.devPtr = (void *)point_value;
 res1Desc.res.linear.desc.f = hipChannelFormatKindFloat;
 res1Desc.res.linear.desc.x = 32; // bits per channel
 res1Desc.res.linear.sizeInBytes =  npoints   *sizeof(cusp::complex<float>);

  hipTextureDesc tex1Desc;
  memset(&tex1Desc, 0, sizeof(tex1Desc));
  tex1Desc.addressMode[0] = hipAddressModeWrap;
  tex1Desc.addressMode[1] = hipAddressModeWrap;
  tex1Desc.readMode = hipReadModeElementType;
  tex1Desc.filterMode = hipFilterModeLinear;
  tex1Desc.normalizedCoords = 0;

  // create texture object: we only have to do this once!
  hipTextureObject_t tex_value=0;
  hipCreateTextureObject(&tex_value, &res1Desc, &tex1Desc, NULL);
  //  }else{

  //  size_t = size_t(-1);
  //  cusp::complex<float> offset = 	cusp::complex<float>(-1);
  //  CUDA_SAFE_CALL(hipBindTexture(&offset,tex_x_int,binned_points));





  //  }
//--------------------------------------------
  */

 //textures
size_t value_tex =  size_t(-1);
hipBindTexture(&value_tex,tex_x_float,point_value);

size_t kernel_tex =  size_t(-1);
hipBindTexture(&kernel_tex,tex_x_float1,kb_table);


            grid_points_cuda_mex_interleaved_kernel<<<grid,block_size>>>( point_value, npoints, grid_size, points_per_bin,
                    bin_dimension_x, bin_dimension_y, binned_points,
                    binned_points_idx, bin_location,
                    binned_points_x, binned_points_y,nbins,
                    kb_table_size,
		    kb_table_scale,kb_table,value_tex,kernel_tex,
  		  grid_value);
            hipDeviceSynchronize();
            
	    // hipDestroyTextureObject(texRef);
 hipDestroyTextureObject(kernel_tex);
 hipDestroyTextureObject(value_tex);


            clock_t t_e = clock();
            error_handle();

        }
        

//#define	SX	    prhs[0]
//#define	SY	prhs[1]
#define	SV	prhs[0]
#define	GRID_DIM	prhs[1]
#define	SPB	prhs[2]
#define	BIN_DIM_X      prhs[3]
#define	BIN_DIM_Y     prhs[4]
#define	SIB     prhs[5]
#define	BSO     prhs[6]
#define	BL     prhs[7]
#define	BPX     prhs[8]
#define	BPY     prhs[9]
#define	KLUT     prhs[10]
#define	KLUTS     prhs[11]


void mexFunction(int nlhs, mxArray * plhs[], int nrhs,const mxArray * prhs[]){

      /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    
    // mxGPUArray const *samples_x;
    // mxGPUArray const *samples_y;
 mxGPUArray const *samples_values;
 mxGPUArray const *samples_per_bin;
 mxGPUArray const *bin_dimensions_x;
 mxGPUArray const *bin_dimensions_y;
 mxGPUArray const *samples_in_bin;
 mxGPUArray const *bin_start_offset;
 mxGPUArray const *bin_location;
 mxGPUArray const *bin_points_x;
 mxGPUArray const *bin_points_y;
 mxGPUArray const *kernel_lookup_table;
//int *grid_dim =(int *) mxGetPr(GRID_DIM);
float kernel_lookup_table_scale = mxGetScalar(KLUTS);

 int *grid_dim0=( int *) (mxGetData(GRID_DIM));

mwSize *grid_dim=(mwSize *)grid_dim0; 

//mexPrintf("Grid Dimensions %d x %d\n",(grid_dim[0]),(grid_dim[1]));
//mexPrintf("Grid Dimensions %d x %d\n",(grid_dim1[0]),(grid_dim1[1]));


// OUTPUT
mxGPUArray  *grid_values;



//samples_x = mxGPUCreateFromMxArray(SX);
//samples_y = mxGPUCreateFromMxArray(SY);
samples_values = mxGPUCreateFromMxArray(SV);
samples_per_bin = mxGPUCreateFromMxArray(SPB);
bin_dimensions_x = mxGPUCreateFromMxArray(BIN_DIM_X);
bin_dimensions_y = mxGPUCreateFromMxArray(BIN_DIM_Y);  
samples_in_bin = mxGPUCreateFromMxArray(SIB);  
bin_start_offset = mxGPUCreateFromMxArray(BSO);  
bin_location = mxGPUCreateFromMxArray(BL);  
bin_points_x = mxGPUCreateFromMxArray(BPX); 
bin_points_y = mxGPUCreateFromMxArray(BPY); 
kernel_lookup_table= mxGPUCreateFromMxArray(KLUT); 

 int nbins = (int) (mxGPUGetNumberOfElements(bin_dimensions_x));
 int npoints =  (int)(mxGPUGetNumberOfElements(bin_points_x));
 int kernel_lookup_table_size = ( int)(mxGPUGetNumberOfElements(kernel_lookup_table));

mwSize ndim= 2;
  
 
// output:
//  float2 * grid_values;
//  float2 * gold_grid_values;
 
//  plhs[0] = jkt_new( grid_dim[0], grid_dim[1], mxSINGLE_CLASS, mxREAL,);

//grid_values= mxGPUCreateGPUArray(ndim,grid_dim,mxSINGLE_CLASS,mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
grid_values= mxGPUCreateGPUArray(ndim, grid_dim,mxSINGLE_CLASS,mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);

// now get the pointer or whatever it is
//const float *d_samples_x = (const float  *)(mxGPUGetDataReadOnly(samples_x));
//const  float *d_samples_y = (const float  *)(mxGPUGetDataReadOnly(samples_y));
// float2 *d_samples_values = (float2  *)(const float2  *)(mxGPUGetDataReadOnly(samples_values));
 const cusp::complex<float> *d_samples_values = (const cusp::complex<float> *)(mxGPUGetDataReadOnly(samples_values));
const  int * d_samples_per_bin = (const int  *)(mxGPUGetDataReadOnly(samples_per_bin));
const  int * d_bin_dimensions_x = (const int  *)(mxGPUGetDataReadOnly(bin_dimensions_x));
const int * d_bin_dimensions_y = (const int  *)(mxGPUGetDataReadOnly(bin_dimensions_y));
const  int * d_samples_in_bin = (const int  *)(mxGPUGetDataReadOnly(samples_in_bin));
const  int * d_bin_start_offset =(const int  *)(mxGPUGetDataReadOnly(bin_start_offset));
const  int * d_bin_location = (const int  *)(mxGPUGetDataReadOnly(bin_location));
const float * d_bin_points_x = (const float  *)(mxGPUGetDataReadOnly(bin_points_x));
const float * d_bin_points_y = (const float  *)(mxGPUGetDataReadOnly(bin_points_y));
float * d_kernel_lookup_table = ( float  *)(mxGPUGetDataReadOnly(kernel_lookup_table));
const uint2 grid_size = {grid_dim[0],grid_dim[1]};

//float2 * d_grid_values = (float2  *)(mxGPUGetData(grid_values));
cusp::complex<float> * d_grid_values = (cusp::complex<float>  *)(mxGPUGetData(grid_values));




grid_points_cuda_interleaved_mex(d_samples_values, npoints, 
				    grid_size, d_samples_per_bin, d_bin_dimensions_x, d_bin_dimensions_y,
				    d_samples_in_bin, d_bin_start_offset, d_bin_location, 
				    d_bin_points_x, d_bin_points_y,
				    nbins, d_kernel_lookup_table,
				    kernel_lookup_table_size,
				    kernel_lookup_table_scale,  d_grid_values);
//mexErrMsgTxt("gpuArray 2");



plhs[0] = mxGPUCreateMxArrayOnGPU(grid_values);

/*
 hipDestroyTextureObject(texRef);
*/


// mxGPUDestroyGPUArray( samples_x);
// mxGPUDestroyGPUArray( samples_y);
 mxGPUDestroyGPUArray( samples_values);
 mxGPUDestroyGPUArray( samples_per_bin);
 mxGPUDestroyGPUArray( bin_dimensions_x);
 mxGPUDestroyGPUArray( bin_dimensions_y);
 mxGPUDestroyGPUArray( samples_in_bin);
 mxGPUDestroyGPUArray( kernel_lookup_table);
 mxGPUDestroyGPUArray( bin_start_offset);
 mxGPUDestroyGPUArray( bin_location);
 mxGPUDestroyGPUArray( bin_points_x);
 mxGPUDestroyGPUArray( bin_points_y);
 mxGPUDestroyGPUArray( grid_values);

}
