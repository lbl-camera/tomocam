#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------------
 * Tomocam Copyright (c) 2018
 *
 * The Regents of the University of California, through Lawrence Berkeley National
 * Laboratory (subject to receipt of any required approvals from the U.S. Dept. of
 *  Energy). All rights reserved.
 *
 * If you have questions about your rights to use or distribute this software,
 * please contact Berkeley Lab's Innovation & Partnerships Office at IPO@lbl.gov.
 *
 * NOTICE. This Software was developed under funding from the U.S. Department of
 * Energy and the U.S. Government consequently retains certain rights. As such, the
 * U.S. Government has been granted for itself and others acting on its behalf a
 * paid-up, nonexclusive, irrevocable, worldwide license in the Software to
 * reproduce, distribute copies to the public, prepare derivative works, and
 * perform publicly and display publicly, and to permit other to do so.
 *---------------------------------------------------------------------------------
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dev_array.h"
#include "utils.cuh"

namespace tomocam {
    const int NX = 1;
    const int NY = 16;
    const int NZ = 16;
    const float MRF_Q = 2.f;
    const float MRF_C = 0.001f;

    __device__ const float FILTER[3][3][3] = {
        {{0.0302, 0.0370, 0.0302}, {0.0370, 0.0523, 0.0370}, {0.0302, 0.0370, 0.0302}},
        {{0.0370, 0.0523, 0.0370}, {0.0532, 0.0000, 0.0523}, {0.0370, 0.0523, 0.0370}},
        {{0.0302, 0.0370, 0.0302}, {0.0370, 0.0523, 0.0370}, {0.0302, 0.0370, 0.0302}}};

    __forceinline__ __device__ float weight(int i, int j, int k) { return FILTER[i][j][k]; }

    /*
     *            (|d| / sigma)^q
     *  f(d) =  -------------------
     *          c + (|d| / sigma)^(q-p)
     */
    __device__ float pot_func(float delta, float MRF_P, float MRF_SIGMA) {
        return ((powf(fabs(delta) / MRF_SIGMA, MRF_Q)) / (MRF_C + powf(fabs(delta) / MRF_SIGMA, MRF_Q - MRF_P)));
    }

    __device__ float deriv_potFCN(float delta, float MRF_P, float MRF_SIGMA) {
        float MRF_SIGMA_Q = powf(MRF_SIGMA, MRF_Q);
        float MRF_SIGMA_Q_P = powf(MRF_SIGMA, MRF_Q - MRF_P);

        float temp1 = powf(fabs(delta), MRF_Q - MRF_P) / MRF_SIGMA_Q_P;
        float temp2 = powf(fabs(delta), MRF_Q - 1);
        float temp3 = MRF_C + temp1;

        if (delta < 0.0) {
            return ((-1 * temp2 / (temp3 * MRF_SIGMA_Q)) * (MRF_Q - ((MRF_Q - MRF_P) * temp1) / (temp3)));
        } else if (delta > 0.0) {
            return ((temp2 / (temp3 * MRF_SIGMA_Q)) * (MRF_Q - ((MRF_Q - MRF_P) * temp1) / (temp3)));
        } else {
            return 0; // MRF_Q / (MRF_SIGMA_Q*MRF_C);
        }
    }

    /*Second Derivative of the potential function at zero */
    __device__ float second_deriv_potFunc_zero(float MRF_SIGMA) {
        float MRF_SIGMA_Q = powf(MRF_SIGMA, MRF_Q);
        return MRF_Q / (MRF_SIGMA_Q * MRF_C);
    }

    __global__ void tvd_update_kernel(DeviceArray<float> model, DeviceArray<float> objfn, float p, float sigma) {

        // thread ids
        int i = threadIdx.x;
        int j = threadIdx.y;
        int k = threadIdx.z;

        // global offsets
        int I0 = blockDim.x * blockIdx.x;
        int J0 = blockDim.y * blockIdx.y;
        int K0 = blockDim.z * blockIdx.z;

        // global ids
        int x = I0 + i;
        int y = J0 + j;
        int z = K0 + k;

        // last thread in the block
        dim3_t dims = objfn.dims();
        int imax = min(dims.x - I0 - 1, blockDim.x - 1);
        int jmax = min(dims.y - J0 - 1, blockDim.y - 1);
        int kmax = min(dims.z - K0 - 1, blockDim.z - 1);

        if ((x < dims.x) && (y < dims.y) && (z < dims.z)) {

            // size of the array
            dim3_t dims = objfn.dims();

            /* copy values into shared memory. */
            __shared__ float s_val[NX + 2][NY + 2][NZ + 2];

            // copy from global memory
            s_val[i + 1][j + 1][k + 1] = model(x, y, z);

            /* copy ghost cells, on all 6 faces */
            // x = 0 face
            if (i == 0) {
                if (x > 0) 
                    s_val[i][j + 1][k + 1] = model(x - 1, y, z);
                else
                    s_val[i][j + 1][k + 1] = 0.f;
            }

            // x = Nx-1 face
            if (i == imax) {
                if (x < dims.x - 1) 
                    s_val[i + 2][j + 1][k + 1] = model(x + 1, y, z);
                else
                    s_val[i + 2][j + 1][k + 1] = 0.f;
            }
            __syncthreads();

            if (j == 0) {
                if (y > 0) 
                    s_val[i + 1][j][k + 1] = model(x, y - 1, z);
                else
                    s_val[i + 1][j][k + 1] = 0.f;
            }

            if (j == jmax) {
                if (y < dims.y - 1) 
                    s_val[i + 1][j + 2][k + 1] = model(x, y + 1, z);
                else
                    s_val[i + 1][j + 2][k + 1] = 0.f;
            }
            __syncthreads();

            if (k == 0) {
                if (z > 0) 
                    s_val[i + 1][j + 1][k] = model(x, y, z - 1);
                else
                    s_val[i + 1][j + 1][k] = 0.f;
            }

            if (k == kmax) {
                if (z < dims.z - 1) 
                    s_val[i + 1][j + 1][k + 2] = model(x, y, z + 1);
                else
                    s_val[i + 1][j + 1][k + 2] = 0.f;
            }
            __syncthreads();

            /* copy ghost cells along 12 edges  */
            if (i == 0) {
                if (j == 0) {
                    if ((x > 0) && (y > 0)) 
                        s_val[i][j][k + 1] = model(x - 1, y - 1, z);
                    else
                        s_val[i][j][k + 1] = 0.f;
                }
                if (j == jmax) {
                    if ((x > 0) && (y < dims.y - 1)) 
                        s_val[i][j + 2][k + 1] = model(x - 1, y + 1, z);
                    else
                        s_val[i][j + 2][k + 1] = 0.f;
                }
            }
            if (i == imax) {
                if (j == 0) {
                    if ((x < dims.x - 1) && (y > 0)) 
                        s_val[i + 2][j][k + 1] = model(x + 1, y - 1, z);
                    else
                        s_val[i + 2][j][k + 1] = 0.f;
                }
                if (j == jmax) {
                    if ((x < dims.x - 1) && (y < dims.y - 1)) 
                        s_val[i + 2][j + 2][k + 1] = model(x + 1, y + 1, z);
                    else
                        s_val[i + 2][j + 2][k + 1] = 0.f;
                }
            }
            __syncthreads();

            if (j == 0) {
                if (k == 0) {
                    if ((y > 0) && (z > 0)) 
                        s_val[i + 1][j][k] = model(x, y - 1, z - 1);
                    else
                        s_val[i + 1][j][k] = 0.f;
                }
                if (k == kmax) {
                    if ((y > 0) && (z < dims.z - 1)) 
                        s_val[i + 1][j][k + 2] = model(x, y - 1, z + 1);
                    else
                        s_val[i + 1][j][k + 2] = 0.f;
                }
            }
            if (j == jmax) {
                if (k == 0) {
                    if ((y < dims.y - 1) && (z > 0)) 
                        s_val[i + 1][j + 2][k] = model(x, y + 1, z - 1);
                    else
                        s_val[i + 1][j + 2][k] = 0.f;
                }
                if (k == kmax) {
                    if ((y < dims.y - 1) && (z < dims.z - 1)) 
                        s_val[i + 1][j + 2][k + 2] = model(x, y + 1, z + 1);
                    else
                        s_val[i + 1][j + 2][k + 2] = 0.f;
                }
            }
            __syncthreads();

            // copy ghost-cells along y-direction
            if (k == 0) {
                if (i == 0) {
                    if ((x > 0) && (z > 0)) 
                        s_val[i][j + 1][k] = model(x - 1, y, z - 1);
                    else
                        s_val[i][j + 1][k] = 0.f;
                }
                if (i == imax) {
                    if ((x < dims.x - 1) && (z > 0)) 
                        s_val[i + 2][j + 1][k] = model(x + 1, y, z - 1);
                    else
                        s_val[i + 2][j + 1][k] = 0.f;
                }
            }
            if (k == kmax) {
                if (i == 0) {
                    if ((x > 0) && (z < dims.z - 1)) 
                        s_val[i][j + 1][k + 2] = model(x - 1, y, z + 1);
                    else
                        s_val[i][j + 1][k + 2] = 0.f;
                }
                if (i == imax) {
                    if ((x < dims.x - 1) && (z < dims.z - 1)) 
                        s_val[i + 2][j + 1][k + 2] = model(x + 1, y, z + 1);
                    else
                        s_val[i + 2][j + 1][k + 2] = 0.f;
                }
            }
            __syncthreads();

            /*  copy  ghost cells along 16 corners */
            if (k == 0) {
                if (j == 0) {
                    if (i == 0) {
                        if ((x > 0) && (y > 0) && (z > 0)) 
                            s_val[i][j][k] = model(x - 1, y - 1, z - 1);
                        else
                            s_val[i][j][k] = 0.f;
                    }
                    if (i == imax) {
                        if ((x < dims.x - 1) && (y > 0) && (z > 0)) 
                            s_val[i + 2][j][k] = model(x + 1, y - 1, z - 1);
                        else
                            s_val[i + 2][j][k] = 0.f;
                    }
                }
                if (j == jmax) {
                    if (i == 0) {
                        if ((x > 0) && (y < dims.y - 1) && (z > 0)) 
                            s_val[i][j + 2][k] = model(x - 1, y + 1, z - 1);
                        else
                            s_val[i][j + 2][k] = 0.f;
                    }
                    if (i == imax) {
                        if ((x < dims.x - 1) && (y < dims.y - 1) && (z > 0))
                            s_val[i + 2][j + 2][k] = model(x + 1, y + 1, z - 1);
                        else
                            s_val[i + 2][j + 2][k] = 0.f;
                    }
                }
            }
            if (k == kmax) {
                if (j == 0) {
                    if (i == 0) {
                        if ((x > 0) && (y > 0) && (z < dims.z - 1)) 
                            s_val[i][j][k + 2] = model(x - 1, y - 1, z + 1);
                        else
                            s_val[i][j][k + 2] = 0.f;
                    }
                    if (i == imax) {
                        if ((x < dims.x - 1) && (y > 0) && (z < dims.z - 1))
                            s_val[i + 2][j][k + 2] = model(x + 1, y - 1, z + 1);
                        else
                            s_val[i + 2][j][k + 2] = 0.f;
                    }
                }
                if (j == jmax) {
                    if (i == 0) {
                        if ((x > 0) && (y < dims.y - 1) && (z < dims.z - 1))
                            s_val[i][j + 2][k + 2] = model(x - 1, y + 1, z + 1);
                        else
                            s_val[i][j + 2][k + 2] = 0.f;
                    }
                    if (i == imax) {
                        if ((x < dims.x - 1) && (y < dims.y - 1) && (z < dims.z - 1))
                            s_val[i + 2][j + 2][k + 2] = model(x + 1, y + 1, z + 1);
                        else
                            s_val[i + 2][j + 2][k + 2] = 0.f;
                    }
                }
            }
            __syncthreads();

            float v = s_val[i + 1][j + 1][i + 1];
            float temp = 0.f;
            for (int iz = 0; iz < 3; iz++)
                for (int iy = 0; iy < 3; iy++)
                    for (int ix = 0; ix < 3; ix++)
                        temp += weight(ix, iy, iz) * deriv_potFCN(v - s_val[i + ix][j + iy][k + iz], p, sigma);
            objfn(x, y, z) += temp;
        }
    }

    __global__ void hessian_zero_kernel(DeviceArray<float> hessian, float sigma) {

        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;
        int z = blockDim.z * blockIdx.z + threadIdx.z;

        dim3_t dims = hessian.dims();
        if ((x < dims.x) && (y < dims.y) && (z < dims.z)) {
            float temp = 0.f;
            for (int ix = 0; ix < 3; ix++)
                for (int iy = 0; iy < 3; iy++)
                    for (int iz = 0; iz < 3; iz++) temp += weight(ix, iy, iz) * second_deriv_potFunc_zero(sigma);
            hessian(x, y, z) += temp;
        }
    }

    void add_total_var(DeviceArray<float> model, DeviceArray<float> objfn, float p, float sigma, hipStream_t stream) {

        // block dims
        dim3 threads(NX, NY, NZ);
        dim3 tblocks = calcBlocks(objfn.dims(), threads);
        tvd_update_kernel<<<tblocks, threads, 0, stream>>>(model, objfn, p, sigma);
    }

    void calcHessian(DeviceArray<float> hessian, float sigma, hipStream_t stream) {

        // block dims
        dim3 threads(NX, NY, NZ);
        dim3 tblocks = calcBlocks(hessian.dims(), threads);

        // update hessain inplace
        hessian_zero_kernel<<<tblocks, threads, 0, stream>>>(hessian, sigma);
    }

} // namespace tomocam
