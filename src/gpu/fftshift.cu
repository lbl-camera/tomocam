#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------------
 * Tomocam Copyright (c) 2018
 *
 * The Regents of the University of California, through Lawrence Berkeley
 *National Laboratory (subject to receipt of any required approvals from the
 *U.S. Dept. of Energy). All rights reserved.
 *
 * If you have questions about your rights to use or distribute this software,
 * please contact Berkeley Lab's Innovation & Partnerships Office at
 *IPO@lbl.gov.
 *
 * NOTICE. This Software was developed under funding from the U.S. Department of
 * Energy and the U.S. Government consequently retains certain rights. As such,
 *the U.S. Government has been granted for itself and others acting on its
 *behalf a paid-up, nonexclusive, irrevocable, worldwide license in the Software
 *to reproduce, distribute copies to the public, prepare derivative works, and
 * perform publicly and display publicly, and to permit other to do so.
 *---------------------------------------------------------------------------------
 */

#include "gpu/dev_memory.cuh"
#include "gpu/utils.cuh"

#include "dev_array.h"

namespace tomocam {
    namespace gpu {

        template <typename T>
        __global__ void roll_kernel(const DeviceMemory<T> in, DeviceMemory<T> out, int delta) {

            // indices
            dim3_t dims = in.dims();
            int3 idx = Index3D();
            if (idx < dims) {
                int3 idx2 = idx;
                idx2.z = (idx.z + delta + dims.z) % dims.z;
                out[idx] = in[idx2];
            }
        }

        template <typename T>
        DeviceArray<T> roll(const DeviceArray<T> &arr, int delta) {

            auto dims = arr.dims();
            DeviceArray<T> out(dims);
            Grid grid(dims);
            roll_kernel<T> <<<grid.blocks(), grid.threads()>>>(arr, out, delta);
            return out;
        }
        // explicit instantiation
        template DeviceArray<float> roll(const DeviceArray<float> &, int);
        template DeviceArray<double> roll(const DeviceArray<double> &, int);
        template DeviceArray<complex_t<float>> roll(
            const DeviceArray<complex_t<float>> &, int);
        template DeviceArray<complex_t<double>> roll(
            const DeviceArray<complex_t<double>> &, int);

        /* -------------------------------------------------------------------- */

        template <typename T>
        __global__ void roll2(const DeviceMemory<T> in, DeviceMemory<T> out, int delta_y, int delta_z) {

            // indices
            dim3_t dims = in.dims();
            int3 idx = Index3D();
            if (idx < dims) {
                int3 idx2 = idx;
                idx2.y = (idx.y + delta_y + dims.y) % dims.y;
                idx2.z = (idx.z + delta_z + dims.z) % dims.z;
                out[idx] = in[idx2];
            }
        }

        template <typename T>
        DeviceArray<T> roll2(const DeviceArray<T> &arr, int delta_y, int delta_z) {

            // indices
            auto dims = arr.dims();
            DeviceArray<T> out(dims);
            Grid grid(dims);
            roll2<T> <<<grid.blocks(), grid.threads()>>>(arr, out, delta_y, delta_z);
            return out;
        }

        // explicit instantiation
        template DeviceArray<float> roll2(const DeviceArray<float> &, int, int);
        template DeviceArray<double> roll2(const DeviceArray<double> &, int, int);
        template DeviceArray<complex_t<float>> roll2(
            const DeviceArray<complex_t<float>> &, int, int);
        template DeviceArray<complex_t<double>> roll2(
            const DeviceArray<complex_t<double>> &, int, int);

    } // namespace gpu
} // namespace tomocam
