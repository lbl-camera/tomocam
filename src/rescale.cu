#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------------
 * Tomocam Copyright (c) 2018
 *
 * The Regents of the University of California, through Lawrence Berkeley
 *National Laboratory (subject to receipt of any required approvals from the
 *U.S. Dept. of Energy). All rights reserved.
 *
 * If you have questions about your rights to use or distribute this software,
 * please contact Berkeley Lab's Innovation & Partnerships Office at
 *IPO@lbl.gov.
 *
 * NOTICE. This Software was developed under funding from the U.S. Department of
 * Energy and the U.S. Government consequently retains certain rights. As such,
 *the U.S. Government has been granted for itself and others acting on its
 *behalf a paid-up, nonexclusive, irrevocable, worldwide license in the Software
 *to reproduce, distribute copies to the public, prepare derivative works, and
 * perform publicly and display publicly, and to permit other to do so.
 *---------------------------------------------------------------------------------
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dev_array.h"
#include "types.h"
#include "utils.cuh"

namespace tomocam {
    __global__ void rescale_kernel(DeviceArray<cuComplex_t> arr, float scale) {
        int3 idx = Index3D();
        if (idx < arr.dims()) 
            arr[idx] = arr[idx]*scale;
    }

    void rescale(dev_arrayc arr, hipStream_t stream) {
        dim3_t dims = arr.dims();
        float scale = 1.f / (dims.z * dims.z);
        Grid grid(dims); 
        rescale_kernel <<< grid.blocks(), grid.threads(), 0, stream >>> (arr, scale);
    }
} // namespace tomocam
