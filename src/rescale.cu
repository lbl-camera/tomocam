#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------------
 * Tomocam Copyright (c) 2018
 *
 * The Regents of the University of California, through Lawrence Berkeley
 *National Laboratory (subject to receipt of any required approvals from the
 *U.S. Dept. of Energy). All rights reserved.
 *
 * If you have questions about your rights to use or distribute this software,
 * please contact Berkeley Lab's Innovation & Partnerships Office at
 *IPO@lbl.gov.
 *
 * NOTICE. This Software was developed under funding from the U.S. Department of
 * Energy and the U.S. Government consequently retains certain rights. As such,
 *the U.S. Government has been granted for itself and others acting on its
 *behalf a paid-up, nonexclusive, irrevocable, worldwide license in the Software
 *to reproduce, distribute copies to the public, prepare derivative works, and
 * perform publicly and display publicly, and to permit other to do so.
 *---------------------------------------------------------------------------------
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dev_array.h"
#include "types.h"

namespace tomocam {
    __global__ void rescale_kernel(cuComplex_t * arr, float scale, int len) {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < len) {
            arr[i].x *= scale;
            arr[i].y *= scale;
        } 
    }

    void rescale(cuComplex_t * arr, dim3_t dims, float scale, hipStream_t stream) {
        int len = dims.x * dims.y * dims.z; 
        dim3 threads(256);
        dim3 tblocks(len / threads.x + 1);
        rescale_kernel <<< tblocks, threads, 0, stream >>> (arr, scale, len);
    }
} // namespace tomocam
