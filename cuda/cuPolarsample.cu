#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "polarsample.h"

texture<float, 1, hipReadModeElementType> texRef;

__device__ float kb_weight1(float grid_pos, float point_pos,
				    int kb_table_size,
				    float kb_table_scale){
  float dist_x = fabs(grid_pos-point_pos)*kb_table_scale;
  int ix = (int)dist_x;
  float fx = dist_x-ix;
  if(ix+1 < kb_table_size){
    return (tex1Dfetch(texRef,ix)*(1.0f-fx) + tex1Dfetch(texRef,ix+1)*(fx));     
  }
  return 0.0f;
}

__global__ void polarsample_kernel(complex_t * point_pos,
				   complex_t * grid_value, 
				   int npoints, uint2 grid_size,
				   int kb_table_size,
				   float kb_table_scale,
				   float kernel_radius,
				    complex_t * sample_value){
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < npoints){

        // initialize sample values
        complex_t sv= make_hipFloatComplex(0.f, 0.f);

        // point positions
        float sx = point_pos[i].x;
        float sy = point_pos[i].y;

        // y-coord at the grid position in nbhd of current point
        int y = max(0, (int) ceil(sy - kernel_radius));
        int ymax = min((int) floor(sy + kernel_radius), (grid_size.y - 1));


        for ( ; y < ymax; y++ ) {
            if (y < 0 || y > grid_size.y-1) continue; 

            // calculate weight at y-coord
            float  kby = kb_weight1((float) y, sy, kb_table_size, kb_table_scale);
 
            // y-coord at the grid position in nbhd of current point
            int x = max(0, (int) ceil(sx - kernel_radius));
            int xmax = min((int) floor(sx + kernel_radius), (grid_size.x - 1));
            for(; x < xmax; x++) {
                float kbx = kb_weight1((float) x, sx, kb_table_size, kb_table_scale);
	            sv = sv + grid_value[y*grid_size.x+x]* kby * kbx;
            }
        }
        sample_value[i] = sv;
    }
}

void polarsample(complex_t * point_pos,
		 complex_t * grid_value, int npoints, 
		 uint2 grid_size,
		 float * kb_table,
		 int kb_table_size,
		 float kb_table_scale,
		 float kernel_radius, 
		 complex_t * sample_value){

  size_t offset;
  hipMemset(sample_value,0,sizeof( complex_t)*npoints);
  hipBindTexture(&offset,texRef, kb_table, sizeof(float)*kb_table_size);
  if(offset != 0){
    fprintf(stderr, "Table not allocated with hipMalloc!\n");
    return;
  }

  int block_size = BLOCKSIZE;
  int grid = (npoints+block_size-1)/block_size;
  clock_t t_i = clock();
    polarsample_kernel<<<grid,block_size>>>(
            point_pos,
		    grid_value, npoints, 
		    grid_size,
		    kb_table_size,
		    kb_table_scale,
		    kernel_radius,
		    sample_value);
  hipDeviceSynchronize();
  clock_t t_e = clock();
  error_handle();
}
