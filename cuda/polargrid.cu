#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <algorithm>
//#include "cutil.h"
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <cusp/complex.h>
#include <cusp/blas/blas.h>
#include "gpu/mxGPUArray.h"
//#include "mxGPUArray.h"
#include "polargrid.h"

float cpu_kb_weight(float2 grid_pos, float2 point_pos, float *kb_table,
                    int kb_table_size, float kb_table_scale) {
    float dist_x = fabs(grid_pos.x - point_pos.x) * kb_table_scale;
    float dist_y = fabs(grid_pos.y - point_pos.y) * kb_table_scale;
    int ix = (int)dist_x;
    float fx = dist_x - ix;
    int iy = (int)dist_y;
    float fy = dist_y - iy;

    if (ix + 1 < kb_table_size && iy + 1 < kb_table_size) {
        return (kb_table[ix] * (1.0f - fx) + kb_table[ix + 1] * (fx))
               * (kb_table[iy] * (1.0f - fy) + kb_table[iy + 1] * (fy));
    }
    return 0.0f;
}

void grid_points_gold(const float *d_point_pos_x, const float *d_point_pos_y,
                      const cusp::complex<float> *d_point_value,
                      const int npoints, const uint2 grid_size,
                      const int *d_points_per_bin, const int *d_binned_points,
                      const int *d_binned_points_idx,
                      const int *d_bin_location, const int *d_bin_dim_x,
                      const int *d_bin_dim_y, int nbins,
                      const float *d_kb_table, int kb_table_size,
                      float kb_table_scale,
                      cusp::complex<float> *d_grid_value) {

    /* we're gonna receive almost all device pointers that we have to convert
     * to CPU memory */

    float *point_pos_x = new float[npoints];
    hipMemcpy(point_pos_x, d_point_pos_x, sizeof(float) * npoints,
               hipMemcpyDeviceToHost);
    float *point_pos_y = new float[npoints];
    hipMemcpy(point_pos_y, d_point_pos_y, sizeof(float) * npoints,
               hipMemcpyDeviceToHost);

    cusp::complex<float> *point_value = new cusp::complex<float>[npoints];
    hipMemcpy(point_value, d_point_value,
               sizeof(cusp::complex<float>) * npoints, hipMemcpyDeviceToHost);
    int *points_per_bin = new int[nbins];
    hipMemcpy(points_per_bin, d_points_per_bin, sizeof(int) * nbins,
               hipMemcpyDeviceToHost);

    int *binned_points_idx = new int[nbins];
    hipMemcpy(binned_points_idx, d_binned_points_idx, sizeof(int) * nbins,
               hipMemcpyDeviceToHost);

    int total_size = 0;
    for (int i = 0; i < nbins; i++) {
        total_size += points_per_bin[i];
        total_size = 32 * ((total_size + 31) / 32);
    }
    int *binned_points = new int[total_size];
    hipMemcpy(binned_points, d_binned_points, sizeof(int) * total_size,
               hipMemcpyDeviceToHost);

    int *bin_location = new int[nbins];
    hipMemcpy(bin_location, d_bin_location, sizeof(int) * nbins,
               hipMemcpyDeviceToHost);

    int *bin_dim_x = new int[nbins];
    hipMemcpy(bin_dim_x, d_bin_dim_x, sizeof(int) * nbins,
               hipMemcpyDeviceToHost);

    int *bin_dim_y = new int[nbins];
    hipMemcpy(bin_dim_y, d_bin_dim_y, sizeof(int) * nbins,
               hipMemcpyDeviceToHost);

    cusp::complex<float> *grid_value = new cusp::complex
        <float>[grid_size.x * grid_size.y];

    memset(grid_value, 0,
           sizeof(cusp::complex<float>) * grid_size.x * grid_size.y);
    float *kb_table = new float[kb_table_size];
    hipMemcpy(kb_table, d_kb_table, sizeof(float) * kb_table_size,
               hipMemcpyDeviceToHost);

    for (int i = 0; i < nbins; i++) {
        uint2 corner;
        corner.x = bin_location[i] % grid_size.x;
        corner.y = bin_location[i] / grid_size.x;
        int idx = binned_points_idx[i];
        for (int y = corner.y; y < corner.y + bin_dim_y[i]; y++) {
            for (int x = corner.x; x < corner.x + bin_dim_x[i]; x++) {
                grid_value[y * grid_size.x + x] = 0;
                for (int j = 0; j < points_per_bin[i]; j++) {
                    grid_value[y * grid_size.x + x]
                        += point_value[binned_points[idx + j]]
                           * cpu_kb_weight(
                                 make_float2(x, y),
                                 make_float2(
                                     point_pos_x[binned_points[idx + j]],
                                     point_pos_y[binned_points[idx + j]]),
                                 kb_table, kb_table_size, kb_table_scale);
                }
            }
        }
    }

    hipMemcpy(d_grid_value, grid_value,
               sizeof(cusp::complex<float>) * grid_size.x * grid_size.y,
               hipMemcpyHostToDevice);
}

//---------
texture<float, 1, hipReadModeElementType> texRef;
texture<int, 1> tex_x_int;
texture<float, 1> tex_x_float;
texture<float, 1> tex_x_float1;

__inline__ __device__ cusp::complex<float>
fetch_x(const int &i, const cusp::complex<float> *x) {
    return cusp::complex<float>(tex1Dfetch(tex_x_float, i * 2),
                                tex1Dfetch(tex_x_float, i * 2 + 1));
}

__inline__ __device__ float fetch_x(const int &i, const float *x) {
    return tex1Dfetch(tex_x_float1, i);
}

void error_handle(hipError_t status = hipErrorLaunchFailure);

void error_handle(hipError_t status) {
    if (status != hipSuccess) {
        hipError_t s = hipGetLastError();
        if (s != hipSuccess) {
            //  printf("%s\n",hipGetErrorString(s));
            exit(1);
        }
    }
}

__device__ float kb_weight(float2 grid_pos, float2 point_pos,
                           int kb_table_size, float kb_table_scale,
                           const float *kb_table) {
    float dist_x = fabsf(grid_pos.x - point_pos.x) * kb_table_scale;
    float dist_y = fabsf(grid_pos.y - point_pos.y) * kb_table_scale;
    // float w=tex1D<float>(texRef,0.0f);

    //    return w;//tex1D<float>(texRef,dist_y);//
    // *tex1D<float>(texRef,dist_y);

    //      return 1.0f;

    int ix = (int)dist_x;
    float fx = dist_x - ix;
    int iy = (int)dist_y;
    float fy = dist_y - iy;

    if (ix + 1 < kb_table_size && iy + 1 < kb_table_size) {
        return (fetch_x(ix, kb_table) * (1.0f - fx) + fetch_x(ix + 1, kb_table) 
               * (fx)) * (fetch_x(iy, kb_table) * (1.0f - fy)
                  + fetch_x(iy + 1, kb_table) * (fy));
    }
    return 0.0f;
}

__device__ float kb_weight(float grid_x, float grid_y, float point_pos_x,
                           float point_pos_y, int kb_table_size,
                           float kb_table_scale, const float *kb_table) {
    float dist_x = fabsf(grid_x - point_pos_x) * kb_table_scale;
    float dist_y = fabsf(grid_y - point_pos_y) * kb_table_scale;

    int ix = (int)dist_x;
    float fx = dist_x - ix;
    int iy = (int)dist_y;
    float fy = dist_y - iy;

    if (ix + 1 < kb_table_size && iy + 1 < kb_table_size) {
        return (fetch_x(ix, kb_table) * (1.0f - fx) + fetch_x(ix + 1, kb_table)
                                                      * (fx))
               * (fetch_x(iy, kb_table) * (1.0f - fy)
                  + fetch_x(iy + 1, kb_table) * (fy));
    }
    return 0.0f;
}

__device__ float kb_weight(float2 grid_pos, float2 point_pos,
                           int kb_table_size, float kb_table_scale,
                           const float *kb_table, int tid) {
    float dist_x = fabsf(grid_pos.x - point_pos.x) * kb_table_scale;
    float dist_y = fabsf(grid_pos.y - point_pos.y) * kb_table_scale;
    float ix = rintf(dist_x);
    float fx = dist_x - ix;
    float iy = rintf(dist_y);
    float fy = dist_y - iy;

    if (ix + 1 < kb_table_size && iy + 1 < kb_table_size) {
        return (fetch_x(ix, kb_table) * (1.0f - fx) + fetch_x(ix + 1, kb_table)
               * (fetch_x(iy, kb_table) * (1.0f - fy)
                  + fetch_x(iy + 1, kb_table) * (fy));
    }
    return 0.0f;
}

//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
__global__ void
sum_points(const cusp::complex<float> *point_value, int npoints,
           uint2 grid_size, const int *points_per_bin,
           const int *bin_dimension_x, const int *bin_dimension_y,
           const int *binned_points, const int *binned_points_idx,
           const int *bin_location, const float *binned_points_x,
           const float *binned_points_y, const int nbins,
           const int kb_table_size, const float kb_table_scale,
           const float *kb_table, hipTextureObject_t texRef,
           cusp::complex<float> *grid_value, int pbid) {
    __shared__ cusp::complex<float> value;

    __shared__ cusp::complex<float> sum_t[BLOCKSIZE];

    int i = pbid;
    int tid = threadIdx.x;
    int jj = blockIdx.x;

    uint2 corner;
    corner.x = bin_location[i] % grid_size.x;
    corner.y = bin_location[i] / grid_size.x;
    const int idx = binned_points_idx[i];
    const int ppb = points_per_bin[i];
    const int bd = BLOCKSIZE;
    int bdx = bin_dimension_x[i];

    //    loop through grid
    for (int yi = corner.y; yi < corner.y + bin_dimension_x[i]; yi += 1) {
        int y = (yi - corner.y + jj) % bdx
                + corner.y; // shift so that there is no overlap

        for (int x = corner.x; x < corner.x + bin_dimension_y[i]; x += 1) {

            sum_t[tid] = 0;

            for (int j = tid + jj * bd; j < ppb; j += bd * gridDim.x) {
                sum_t[tid]
                    += point_value[binned_points[idx + j]]
                       * kb_weight(make_float2(x, y),
                                   make_float2(binned_points_x[idx + j],
                                               binned_points_y[idx + j]),
                                   kb_table_size, kb_table_scale, kb_table,
                                   texRef);
            }
            __syncthreads();

            for (unsigned int j = 1; j < bd; j *= 2) {
                // modulo arithmetic is slow!
                if ((tid & (2 * j - 1)) == 0) {
                    sum_t[tid] += sum_t[tid + j];
                }
                __syncthreads();
            }

            if (tid == 0) {
                atomicAdd(&(grid_value[y * grid_size.x + x]).x, (sum_t[0]).x);
                atomicAdd(&(grid_value[y * grid_size.x + x]).y, (sum_t[0]).y);
            }
        }
    }
}


//--------------------------------
__global__ void grid_points_cuda_mex_interleaved_kernel1(
    const float *point_x, const float *point_y,
    const cusp::complex<float> *point_value, int npoints, uint2 grid_size,
    const int *points_per_bin, const int *bin_dimension_x,
    const int *bin_dimension_y, const int *binned_points,
    const int *binned_points_idx, const int *bin_location,
    const float *binned_points_x, const float *binned_points_y, int nbins,
    int kb_table_size, float kb_table_scale, const float *kb_table,
    cusp::complex<float> *grid_value) {

    __shared__ float point_pos_cache_x[SHARED_SIZE];
    __shared__ float point_pos_cache_y[SHARED_SIZE];
    __shared__ cusp::complex<float> point_value_cache[SHARED_SIZE];
    __shared__ cusp::complex<float> sum_t[BLOCKSIZE];

    int i = blockIdx.x;
    int tid = threadIdx.x;
    uint2 corner;
    corner.x = bin_location[i] % grid_size.x;
    corner.y = bin_location[i] / grid_size.x;
    int idx = binned_points_idx[i];
    const int ppb = points_per_bin[i];

    // small bin or large no of samples
    if (bin_dimension_x[i] * bin_dimension_y[i] < 64 || points_per_bin[i]
                                                        > SHARED_SIZE) {
        //    loop through grid
        for (int y = corner.y; y < corner.y + bin_dimension_x[i]; y += 1) {
            for (int x = corner.x; x < corner.x + bin_dimension_y[i]; x += 1) {
                sum_t[tid] = 0;
                // loop through points
                for (int j = tid; j < ppb; j += blockDim.x) {
                    sum_t[tid]
                        += point_value[binned_points[idx + j]]
                           * kb_weight(make_float2(x, y),
                                       make_float2(binned_points_x[idx + j],
                                                   binned_points_y[idx + j]),
                                       kb_table_size, kb_table_scale,
                                       kb_table);
                }
                // Do a reduce in shared memory
                for (unsigned int j = 1; j < blockDim.x; j *= 2) {
                    // modulo arithmetic is slow!
                    if ((tid & (2 * j - 1)) == 0) {
                        sum_t[tid] += sum_t[tid + j];
                    }
                    __syncthreads();
                }
                if (tid == 0) {
                    grid_value[y * grid_size.x + x] = sum_t[0];
                }
            }
        }
        // large dimensions
    } else if (bin_dimension_x[i] * bin_dimension_y[i] > BLOCKSIZE / 2 - 1) {
        /* Lets try to load all points to shared memory */
        const int ppb = points_per_bin[i];
        for (int j = tid; j < ppb; j += blockDim.x) {
            const int point = binned_points[idx + j];
            point_value_cache[j] = point_value[point];
            point_pos_cache_x[j] = binned_points_x[idx + j];
            point_pos_cache_y[j] = binned_points_y[idx + j];
        }
        __syncthreads();
        const uint2 dims = { bin_dimension_x[i], bin_dimension_y[i] };
        // loop through dimensions
        for (int k = tid; k < dims.x * dims.y; k += blockDim.x) {
            const int x = (k % (dims.x)) + corner.x;
            const int y = (k / dims.x) + corner.y;
            cusp::complex<float> my_sum = 0;
            for (int j = 0; j < ppb; j++) { // loop through all the points
                float w = kb_weight(x, y, point_pos_cache_x[j],
                                    point_pos_cache_y[j], kb_table_size,
                                    kb_table_scale, kb_table);
                my_sum += point_value_cache[j] * w;
            }
            grid_value[y * grid_size.x + x] = my_sum;
        }
    } else { // small dimension and few points
        /* Lets try to load things to shared memory */
        const int ppb = points_per_bin[i];
        for (int j = tid; j < ppb; j += blockDim.x) {
            const int point = binned_points[idx + j];
            point_value_cache[j] = point_value[point];
            point_pos_cache_x[j] = binned_points_x[idx + j];
            point_pos_cache_y[j] = binned_points_y[idx + j];
        }
        __syncthreads();
        const uint2 dims = { bin_dimension_x[i], bin_dimension_y[i] };
        int b = 4;
        for (int k = tid / b; k < dims.x * dims.y; k += blockDim.x / b) {
            const int x = (k % (dims.x)) + corner.x;
            const int y = (k / dims.x) + corner.y;
            sum_t[tid] = 0;
            // sum_i[tid] = 0;
            for (int j = (tid & (b - 1)); j < ppb; j += b) {
                float w = kb_weight(x, y, point_pos_cache_x[j],
                                    point_pos_cache_y[j], kb_table_size,
                                    kb_table_scale, kb_table);
                sum_t[tid] += point_value_cache[j] * w;
            }
            /* Do a reduce in shared memory */
            for (unsigned int j = 1; j < b; j = (j << 1)) {
                // modulo arithmetic is slow!
                if ((tid & ((j << 1) - 1)) == 0) {
                    sum_t[tid] += sum_t[tid + j];
                }
                __syncthreads();
            }
            if ((tid & (b - 1)) == 0) {
                grid_value[y * grid_size.x + x] = sum_t[tid];
            }
        }
    }
}

void grid_points_cuda_interleaved_mex(
    const float *point_pos_x, const float *point_pos_y,
    const cusp::complex<float> *point_value, int npoints, uint2 grid_size,
    const int *points_per_bin, const int *bin_dimension_x,
    const int *bin_dimension_y, const int *binned_points,
    const int *binned_points_idx, const int *bin_location,
    const float *binned_points_x, const float *binned_points_y, int nbins,
    const float *kb_table, int kb_table_size, float kb_table_scale,
    cusp::complex<float> *grid_value) {
    hipMemset(grid_value, 0, sizeof(float2) * grid_size.x * grid_size.y);

    size_t offset;
    hipBindTexture(&offset, texRef, kb_table, sizeof(float) * kb_table_size);
    if (offset != 0) {
        //   printf("Error: Texture offset different than zero. Table not
        // allocated with hipMalloc!%d\n");
        return;
    }

    int grid = nbins;
    int block_size = BLOCKSIZE;
    clock_t t_i = clock();
    grid_points_cuda_mex_interleaved_kernel1 << <grid, block_size>>
        > (point_pos_x, point_pos_y, point_value, npoints, grid_size,
           points_per_bin, bin_dimension_x, bin_dimension_y, binned_points,
           binned_points_idx, bin_location, binned_points_x, binned_points_y,
           nbins, kb_table_size, kb_table_scale, kb_table, grid_value);
    hipDeviceSynchronize();

    clock_t t_e = clock();
    error_handle();
    //  printf("%d iter in %5.1f ms\n",iter,(t_e-t_i)*1000.0/CLOCKS_PER_SEC);
}

void compare_to_gold(float *gridded, float *gold_gridded, uint2 grid_size) {
    for (int i = 0; i < grid_size.x * grid_size.y; i++) {
        if (fabs(gridded[i] - gold_gridded[i]) / gridded[i] > 1e-5
            && fabs(gridded[i] - gold_gridded[i]) > 1e-7) {
            //      printf("cuda[%d] = %e gold[%d] =
            // %e\n",i,gridded[i],i,gold_gridded[i]);
            exit(1);
        }
    }
}

#define SX prhs[0]
#define SY prhs[1]
#define SV prhs[2]
#define GRID_DIM prhs[3]
#define SPB prhs[4]
#define BIN_DIM_X prhs[5]
#define BIN_DIM_Y prhs[6]
#define SIB prhs[7]
#define BSO prhs[8]
#define BL prhs[9]
#define BPX prhs[10]
#define BPY prhs[11]
#define KLUT prhs[12]
#define KLUTS prhs[13]

static PyObject *polargrid(PyObject *self, PyObject *args) {

    PyObject *in0, *in1, *in2, *in3, *in4, *in5;
    int bin_start_offset, bin_location;

    if(!PyPrase_Tuple("")) {
        fprintf(stderr,"Failed to parse arguments\n");
        return NULL;
    }
    af::array SX = PyAfnumpy_AsArrayFireArray(in0, FLOAT32);
    af::array SY = PyAfnumpy_AsArrayFireArray(in1, FLOAT32);
    af::array SV = PyAfnumpy_AsArrayFireArray(in2, CMPLX32);
    af::array GRID = PyAfnumpy_AsArrayFireArray();

    mxGPUArray const *samples_x;
    mxGPUArray const *samples_y;
    mxGPUArray const *samples_values;
    mxGPUArray const *samples_per_bin;
    mxGPUArray const *bin_dimensions_x;
    mxGPUArray const *bin_dimensions_y;
    mxGPUArray const *samples_in_bin;
    mxGPUArray const *bin_start_offset;
    mxGPUArray const *bin_location;
    mxGPUArray const *bin_points_x;
    mxGPUArray const *bin_points_y;
    mxGPUArray const *kernel_lookup_table;
    float kernel_lookup_table_scale = mxGetScalar(KLUTS);

    int *grid_dim0 = (int *)(mxGetData(GRID_DIM));

    mwSize *grid_dim = (mwSize *)grid_dim0;

    // OUTPUT
    mxGPUArray *grid_values, *gold_grid_values;

    samples_x = mxGPUCreateFromMxArray(SX);
    samples_y = mxGPUCreateFromMxArray(SY);
    samples_values = mxGPUCreateFromMxArray(SV);
    samples_per_bin = mxGPUCreateFromMxArray(SPB);
    bin_dimensions_x = mxGPUCreateFromMxArray(BIN_DIM_X);
    bin_dimensions_y = mxGPUCreateFromMxArray(BIN_DIM_Y);
    samples_in_bin = mxGPUCreateFromMxArray(SIB);
    bin_start_offset = mxGPUCreateFromMxArray(BSO);
    bin_location = mxGPUCreateFromMxArray(BL);
    bin_points_x = mxGPUCreateFromMxArray(BPX);
    bin_points_y = mxGPUCreateFromMxArray(BPY);
    kernel_lookup_table = mxGPUCreateFromMxArray(KLUT);

    int nbins = (int)(mxGPUGetNumberOfElements(bin_dimensions_x));
    int npoints = (int)(mxGPUGetNumberOfElements(samples_x));
    int kernel_lookup_table_size
        = (int)(mxGPUGetNumberOfElements(kernel_lookup_table));

    mwSize ndim = 2;

    // now get the pointer or whatever it is
    const float *d_samples_x
        = (const float *)(mxGPUGetDataReadOnly(samples_x));
    const float *d_samples_y
        = (const float *)(mxGPUGetDataReadOnly(samples_y));
    // float2 *d_samples_values = (float2  *)(const float2
    // *)(mxGPUGetDataReadOnly(samples_values));
    const cusp::complex<float> *d_samples_values
        = (const cusp::complex<float> *)(mxGPUGetDataReadOnly(samples_values));
    const int *d_samples_per_bin
        = (const int *)(mxGPUGetDataReadOnly(samples_per_bin));
    const int *d_bin_dimensions_x
        = (const int *)(mxGPUGetDataReadOnly(bin_dimensions_x));
    const int *d_bin_dimensions_y
        = (const int *)(mxGPUGetDataReadOnly(bin_dimensions_y));
    const int *d_samples_in_bin
        = (const int *)(mxGPUGetDataReadOnly(samples_in_bin));
    const int *d_bin_start_offset
        = (const int *)(mxGPUGetDataReadOnly(bin_start_offset));
    const int *d_bin_location
        = (const int *)(mxGPUGetDataReadOnly(bin_location));
    const float *d_bin_points_x
        = (const float *)(mxGPUGetDataReadOnly(bin_points_x));
    const float *d_bin_points_y
        = (const float *)(mxGPUGetDataReadOnly(bin_points_y));
    const float *d_kernel_lookup_table
        = (const float *)(mxGPUGetDataReadOnly(kernel_lookup_table));
    const uint2 grid_size = { grid_dim[0], grid_dim[1] };

    float * d_samples_x = Sx->device_ptr<float>();
    float * d_samples_y = Sy->device_ptr<float>();
    cusp::complex<float> * d_samples_values = Sy->device_ptr<cusp::complex<float> >();
    

    // float2 * d_grid_values = (float2  *)(mxGPUGetData(grid_values));
    cusp::complex<float> *d_grid_values
        = (cusp::complex<float> *)(mxGPUGetData(grid_values));

    //   mexErrMsgTxt("gpuArray 1");

    grid_points_cuda_interleaved_mex(
        d_samples_x, d_samples_y, d_samples_values, npoints, grid_size,
        d_samples_per_bin, d_bin_dimensions_x, d_bin_dimensions_y,
        d_samples_in_bin, d_bin_start_offset, d_bin_location, d_bin_points_x,
        d_bin_points_y, nbins, d_kernel_lookup_table, kernel_lookup_table_size,
        kernel_lookup_table_scale, d_grid_values);
    // mexErrMsgTxt("gpuArray 2");

    plhs[0] = mxGPUCreateMxArrayOnGPU(grid_values);

    if (nlhs == 2) {
        // gold_grid_values=  mxGPUCreateGPUArray(ndim,
        // grid_dim,mxSINGLE_CLASS,mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
        gold_grid_values
            = mxGPUCreateGPUArray(ndim, grid_dim, mxSINGLE_CLASS, mxCOMPLEX,
                                  MX_GPU_INITIALIZE_VALUES);
        // gold_grid_values=
        // mxGPUCreateGPUArray(ndim,grid_dim,mxSINGLE_CLASS,mxCOMPLEX,
        // MX_GPU_DO_NOT_INITIALIZE);
        // float2 * d_gold_grid_values = (float2
        // *)(mxGPUGetData(gold_grid_values));
        cusp::complex<float> *d_gold_grid_values
            = (cusp::complex<float> *)(mxGPUGetData(gold_grid_values));

        grid_points_gold(d_samples_x, d_samples_y, d_samples_values, npoints,
                         grid_size, d_samples_per_bin, d_samples_in_bin,
                         d_bin_start_offset, d_bin_location,
                         d_bin_dimensions_x, d_bin_dimensions_y, nbins,
                         d_kernel_lookup_table, kernel_lookup_table_size,
                         kernel_lookup_table_scale, d_gold_grid_values);

        plhs[1] = mxGPUCreateMxArrayOnGPU(gold_grid_values);
        mxGPUDestroyGPUArray(gold_grid_values);
    }
    mxGPUDestroyGPUArray(samples_x);
    mxGPUDestroyGPUArray(samples_y);
    mxGPUDestroyGPUArray(samples_values);
    mxGPUDestroyGPUArray(samples_per_bin);
    mxGPUDestroyGPUArray(bin_dimensions_x);
    mxGPUDestroyGPUArray(bin_dimensions_y);
    mxGPUDestroyGPUArray(samples_in_bin);
    mxGPUDestroyGPUArray(kernel_lookup_table);
    mxGPUDestroyGPUArray(bin_start_offset);
    mxGPUDestroyGPUArray(bin_location);
    mxGPUDestroyGPUArray(bin_points_x);
    mxGPUDestroyGPUArray(bin_points_y);
    mxGPUDestroyGPUArray(grid_values);
}
