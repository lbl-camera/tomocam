#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "polarsample.h"

const int DIMX = 16;
const int DIMY = 4;
const int DIMZ = 4;
const int WORK = 8;
const int sNX = DIMX + 2;
const int sNY = DIMY + 2;
const int sNZ = DIMZ + 2;

__device__ const float FILTER[3][3][3] = 
    { 
        {{0.0302, 0.037, 0.0302}, {0.037, 0.0523, 0.037}, {0.0302, 0.037, 0.0302}},
        {{0.037,  0.0523, 0.037}, {0.0532, 0., 0.0523}, {0.037, 0.0523, 0.037}},
        {{0.0302, 0.037, 0.0302}, {0.037, 0.0523, 0.037}, {0.0302, 0.037, 0.0302}}
    };

__constant__ int nx, ny, nz;

__inline__ __device__ float wght(int k, int j, int i){
    return FILTER[k][j][i];
    
}

__inline__ __device__ int globalIdx(int x, int y, int z){
    return (nx * ny * z + nx * y + x);
}

__inline__ __device__ float deriv_potFCN(float delta) {
    float MRF_C = .001;
    float MRF_P = 1.2;
    float MRF_Q = 2;
    float MRF_SIGMA = 1;
    float MRF_SIGMA_Q = pow(MRF_SIGMA,MRF_Q);
    float MRF_SIGMA_Q_P = pow(MRF_SIGMA,MRF_Q - MRF_P);

    float temp1 = pow(fabs(delta), MRF_Q - MRF_P) / MRF_SIGMA_Q_P;
    float temp2 = pow(fabs(delta), MRF_Q - 1);
    float temp3 = MRF_C + temp1;

    if(delta < 0.0) {
        return ((-1*temp2/(temp3*MRF_SIGMA_Q))*(MRF_Q - ((MRF_Q-MRF_P)*temp1)/(temp3)));
    } else if(delta > 0.0) {
        return ((temp2/(temp3*MRF_SIGMA_Q))*(MRF_Q - ((MRF_Q-MRF_P)*temp1)/(temp3)));
    } else {
        return MRF_Q / (MRF_SIGMA_Q*MRF_C);
    }
}


__global__ void tvd_update_kernel(complex_t * val, complex_t * tvd){
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z; 
    int xOffset = blockDim.x * blockIdx.x;
    int yOffset = blockDim.y * blockIdx.y;
    int zOffset = blockDim.z * blockIdx.z;
    int x = i + xOffset;
    int y = j + yOffset;
    int z = k + zOffset;
       
    if ((x < nx) && (y < ny) && (z < nz)) {

        int gid = globalIdx(x, y, z);

        /* copy values into shared memory. 
         * Max size of shared memory = 64 x 1024 Bytes
         * which translates to  8192 complex number
         */

        const complex_t CMPLX_ZERO = make_hipFloatComplex(0.f, 0.f);
        __shared__ complex_t s_val[sNZ][sNY][sNX];

        // copy from global memory
        s_val[k+1][j+1][i+1] = val[gid];

        /* copy ghost cells, except corners */
        if (i == 0){
            if (x > 0) s_val[k][j][i] = val[globalIdx(x-1, y, z)];
            else s_val[k][j][i] = CMPLX_ZERO;
        }

        if (j == 0){
            if (y > 0) s_val[k][j][i] = val[globalIdx(x, y-1, z)];
            else s_val[k][j][i] = CMPLX_ZERO;
        }

        if (k == 0){
            if (z > 0) s_val[k][j][i] = val[globalIdx(x, y, z-1)];
            else s_val[k][j][i] = CMPLX_ZERO;
        }

        int xlen = min(sNX, nx - xOffset);
        if (i == xlen-1) {
            if (xOffset + xlen < nx) s_val[k][j][i+2] = val[gid+1];
            else s_val[k][j][i+2] = CMPLX_ZERO;
        }

        int ylen = min(sNY, ny - yOffset);
        if (j == ylen-1) {
            if (yOffset + ylen < ny) s_val[k][j+2][i] = val[globalIdx(x, y+1,z)];
            else s_val[k][j+2][i] = CMPLX_ZERO;
        }

        int zlen = min(sNZ, nz - zOffset);
        if (k == zlen-1) {
            if (zOffset + zlen < nz) s_val[k+2][j][i] = val[globalIdx(x, y, z+1)];
            else s_val[k+2][j][i] = CMPLX_ZERO;
        }

        __syncthreads();

        /* copy the corners, all eight of them */
        if (k == 0){
            if (j == 0){
                if (i == 0){
                    if ((x > 0) && (y > 0) && (z > 0)) 
                        s_val[k][j][i] = val[globalIdx(x-1,y-1,z-1)];
                    else s_val[k][j][i] = CMPLX_ZERO;
                }
                if (i == xlen-1) {
                    if (xOffset + xlen < nx)
                        s_val[k][j][i+2] = val[globalIdx(x+1, y-1, z-1)];
                    else s_val[k][j][i+2] = CMPLX_ZERO;
                }
            }
            if (j == ylen-1){
                if (i == 0){
                    if ((x > 0) && (yOffset + ylen < ny) && (z > 0))
                        s_val[k][j+2][i] = val[globalIdx(x-1, y+1, z-1)];
                    else s_val[k][j+2][i] = CMPLX_ZERO;
                }
                if (i == xlen-1){
                    if ((xOffset + xlen < nx) && (yOffset + ylen < ny) && (z > 0))
                        s_val[k][j+2][i+2] = val[globalIdx(x+1, y+1, z-1)];
                    else s_val[k][j+2][i+2] = CMPLX_ZERO;
                }
            }
        }
        if (k == zlen-1){
            if (j == 0){
                if (i == 0){
                    if ((x > 0) && (y > 0) && (zOffset + zlen < nz)) 
                        s_val[k+2][j][i] = val[globalIdx(x-1,y-1,z+1)];
                    else s_val[k+2][j][i] = CMPLX_ZERO;
                }
                if (i == xlen-1){
                    if (xOffset + xlen < nx)
                        s_val[k+2][j][i+2] = val[globalIdx(x+1, y-1, z+1)];
                    else s_val[k+2][j][i+2] = CMPLX_ZERO;
                }
            }
            if (j == ylen-1){
                if (i == 0){
                    if ((x > 0) && (yOffset + ylen < ny) && (zOffset + zlen < nz))
                        s_val[k+2][j+2][i] = val[globalIdx(x-1, y+1, z+1)];
                    else s_val[k+2][j+2][i] = CMPLX_ZERO;
                }
                if (i == xlen-1){
                    if ((xOffset + xlen < nx) && (yOffset + ylen < ny) && (zOffset + zlen < nz))
                        s_val[k+2][j+2][i+2] = val[globalIdx(x+1, y+1, z+1)];
                    else s_val[k+2][j+2][i+2] = CMPLX_ZERO;
                }
            }
        }
        __syncthreads();
    
        complex_t v = s_val[k+1][j+1][i+1];
        for (int iy = 0; iy < 3; iy++)
            for (int ix = 0; ix  < 3; ix++) {
                // same slice as current element
                tvd[gid].x += wght(1, iy, ix) * deriv_potFCN(v.x-s_val[k+1][j+iy][i+ix].x);
                tvd[gid].y += wght(1, iy, ix) * deriv_potFCN(v.y-s_val[k+1][j+iy][i+ix].y);

                //  current slice - 1
                tvd[gid].x += wght(0, iy, ix) * deriv_potFCN(v.x-s_val[k][j+iy][i+ix].y);
                tvd[gid].y += wght(0, iy, ix) * deriv_potFCN(v.y-s_val[k+1][j+iy][i+ix].x);

                //  current slice + 1
                tvd[gid].x += wght(2, iy, ix) * deriv_potFCN(v.x-s_val[k+1][j+iy][i+ix].y);
                tvd[gid].y += wght(2, iy, ix) * deriv_potFCN(v.y-s_val[k+2][j+iy][i+ix].x);
            }
    }
}


void addTVD(int nslice, int nrow, int ncol, complex_t * objfn, complex_t * val) {

    //printf("nslice = %d, nrow = %d, ncol = %d\n", nslice, nrow, ncol);
    int GRIDX = ncol % DIMX > 0 ? ncol/DIMX+1 : ncol/DIMX;
    int GRIDY = nrow % DIMY > 0 ? nrow/DIMY+1 : nrow/DIMY;
    int GRIDZ = nslice%DIMZ > 0 ? nslice/DIMZ+1 : nslice/DIMZ;

    // block dims
    dim3 block(DIMX, DIMY, DIMZ);
    dim3 grid(GRIDX, GRIDY, GRIDZ);

#ifdef DEBUG
    fprintf(stderr, "block = (%d, %d, %d)\n", block.x, block.y, block.z);
    fprintf(stderr, "grid = (%d, %d, %d)\n", grid.x, grid.y, grid.z);
#endif // DEBUG

    /* copy the grid dimensions to constant memeory */
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(nx), &ncol, sizeof(int));   error_handle();
    status = hipMemcpyToSymbol(HIP_SYMBOL(ny), &nrow, sizeof(int));   error_handle();
    status = hipMemcpyToSymbol(HIP_SYMBOL(nz), &nslice, sizeof(int)); error_handle();

    tvd_update_kernel<<<grid, block>>> (val, objfn);
    error_handle();

#ifdef DEBUG
    complex_t * f = new complex_t[nrow * ncol];
    hipMemcpy(f, objfn, sizeof(complex_t) * nrow * ncol, hipMemcpyDeviceToHost);
    for (int j = 0; j < nrow; ++j){
        for (int i = 0; i < ncol; ++i){
            printf("%f   ", f[j * nrow + i].x);
        }
        printf("\n");
    }
    delete [] f;
#endif  // DEBUG
}
